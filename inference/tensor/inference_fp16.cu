#include "hip/hip_runtime.h"
#include </usr/include/hipDNN.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <cassert>
#include <algorithm>
#include <chrono>
#include <iomanip>
#include <hip/hip_fp16.h>

// CUDA error checking
#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)
template<typename T>
void check(T err, const char* const func, const char* const file,
           const int line) {
    if (err != hipSuccess) {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                  << std::endl;
        std::cerr << hipGetErrorString(err) << " " << func << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

// Helper kernels for FP32 to FP16 and vice versa
__global__ void convertFP32ToFP16(float* input, half* output, int size) {
   int idx = blockIdx.x * blockDim.x + threadIdx.x;
   if (idx < size) {
       output[idx] = __float2half(input[idx]);
   }
}

__global__ void convertFP16ToFP32(half* input, float* output, int size) {
   int idx = blockIdx.x * blockDim.x + threadIdx.x;
   if (idx < size) {
       output[idx] = __half2float(input[idx]);
   }
}

void convertToFP16(float* input, half* output, int size) {
   int blockSize = 256;
   int numBlocks = (size + blockSize - 1) / blockSize;
   convertFP32ToFP16<<<numBlocks, blockSize>>>(input, output, size);
   CHECK_CUDA_ERROR(hipGetLastError());
}

void convertToFP32(half* input, float* output, int size) {
   int blockSize = 256;
   int numBlocks = (size + blockSize - 1) / blockSize;
   convertFP16ToFP32<<<numBlocks, blockSize>>>(input, output, size);
   CHECK_CUDA_ERROR(hipGetLastError());
}

template <typename T>
std::vector<T> loadBinaryFile(const std::string& filename) {
   std::ifstream file(filename, std::ios::binary);
   if (!file.is_open()) {
       std::cerr << "Error: Could not open file " << filename << std::endl;
       std::exit(EXIT_FAILURE);
   }
   
   file.seekg(0, std::ios::end);
   size_t file_size = file.tellg();
   file.seekg(0, std::ios::beg);
   
   size_t num_elements = file_size / sizeof(T);
   
   std::vector<T> buffer(num_elements);
   file.read(reinterpret_cast<char*>(buffer.data()), file_size);
   file.close();
   
   return buffer;
}

template <>
std::vector<half> loadBinaryFile<half>(const std::string& filename) {
   std::ifstream file(filename, std::ios::binary);
   if (!file.is_open()) {
       std::cerr << "Error: Could not open file " << filename << std::endl;
       std::exit(EXIT_FAILURE);
   }
   file.seekg(0, std::ios::end);
   size_t size = file.tellg() / sizeof(half);
   file.seekg(0, std::ios::beg);
   std::vector<half> buffer(size);
   file.read(reinterpret_cast<char*>(buffer.data()), size * sizeof(half));
   file.close();
   return buffer;
}

#define CUDNN_CHECK(call) { \
   hipdnnStatus_t err = call; \
   if (err != HIPDNN_STATUS_SUCCESS) { \
       std::cerr << "CuDNN Error: " << hipdnnGetErrorString(err) << std::endl; \
       std::exit(EXIT_FAILURE); \
   } \
}

#define CUDA_CHECK(call) { \
   hipError_t err = call; \
   if (err != hipSuccess) { \
       std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl; \
       std::exit(EXIT_FAILURE); \
   } \
}

class TensorCNNInference {
public:
   TensorCNNInference(int batch_size, const std::string& weights_path);
   ~TensorCNNInference();
   void loadWeights(const std::string& weights_path);
   void initializeLayers();
   void checkTensorCoreUsage();
   void infer(const std::vector<float>& input_data);
   std::vector<float> getOutput();

private:
   hipdnnHandle_t cudnn;
   
   hipdnnTensorDescriptor_t input_desc;
   hipdnnTensorDescriptor_t conv1_output_desc;
   hipdnnTensorDescriptor_t pool1_output_desc;
   hipdnnTensorDescriptor_t conv2_output_desc;
   hipdnnTensorDescriptor_t pool2_output_desc;
   hipdnnTensorDescriptor_t pool2_flat_desc;
   hipdnnTensorDescriptor_t fc1_output_desc;
   hipdnnTensorDescriptor_t fc2_output_desc;

   hipdnnFilterDescriptor_t conv1_filter_desc;
   hipdnnFilterDescriptor_t conv2_filter_desc;
   hipdnnFilterDescriptor_t fc1_filter_desc;
   hipdnnFilterDescriptor_t fc2_filter_desc;
   
   hipdnnTensorDescriptor_t conv1_bias_desc;
   hipdnnTensorDescriptor_t conv2_bias_desc;
   hipdnnTensorDescriptor_t fc1_bias_desc;
   hipdnnTensorDescriptor_t fc2_bias_desc;
   
   hipdnnConvolutionDescriptor_t conv1_desc;
   hipdnnConvolutionDescriptor_t conv2_desc;
   hipdnnConvolutionDescriptor_t fc1_desc;
   hipdnnConvolutionDescriptor_t fc2_desc;
   
   hipdnnActivationDescriptor_t relu_activation;
   hipdnnPoolingDescriptor_t pooling_desc;
   
   hipdnnConvolutionFwdAlgo_t conv1_algo;
   hipdnnConvolutionFwdAlgo_t conv2_algo;
   hipdnnConvolutionFwdAlgo_t fc1_algo;
   hipdnnConvolutionFwdAlgo_t fc2_algo;

   int batch_size;

   float *d_input;
   half *d_conv1_weight, *d_conv1_bias, *d_conv1_output;
   half *d_pool1_output;
   half *d_conv2_weight, *d_conv2_bias, *d_conv2_output;
   half *d_pool2_output;
   half *d_fc1_weight, *d_fc1_bias, *d_fc1_output;
   half *d_fc2_weight, *d_fc2_bias, *d_fc2_output;
   
   size_t workspace_size;
   void *d_workspace;

   struct LayerDims {
       int n, c, h, w;
   };
       LayerDims input_dims, conv1_dims, pool1_dims, conv2_dims, pool2_dims, 
              fc1_dims, fc2_dims;
   hipdnnConvolutionFwdAlgo_t findBestConvAlgorithm(
       hipdnnTensorDescriptor_t input_desc,
       hipdnnFilterDescriptor_t filter_desc,
       hipdnnConvolutionDescriptor_t conv_desc,
       hipdnnTensorDescriptor_t output_desc,
       size_t* workspace_size);
};

TensorCNNInference::TensorCNNInference(int batch_size_, const std::string& weights_path)
    : batch_size(batch_size_) {
    std::cout << "Initializing TensorCore CNN..." << std::endl;
    
    // Create cuDNN handle first
    CUDNN_CHECK(hipdnnCreate(&cudnn));
    
    // Create descriptors
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&input_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&conv1_output_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&pool1_output_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&conv2_output_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&pool2_output_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&pool2_flat_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&fc1_output_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&fc2_output_desc));

    CUDNN_CHECK(hipdnnCreateFilterDescriptor(&conv1_filter_desc));
    CUDNN_CHECK(hipdnnCreateFilterDescriptor(&conv2_filter_desc));
    CUDNN_CHECK(hipdnnCreateFilterDescriptor(&fc1_filter_desc));
    CUDNN_CHECK(hipdnnCreateFilterDescriptor(&fc2_filter_desc));

    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&conv1_bias_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&conv2_bias_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&fc1_bias_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&fc2_bias_desc));

    CUDNN_CHECK(hipdnnCreateConvolutionDescriptor(&conv1_desc));
    CUDNN_CHECK(hipdnnCreateConvolutionDescriptor(&conv2_desc));
    CUDNN_CHECK(hipdnnCreateConvolutionDescriptor(&fc1_desc));
    CUDNN_CHECK(hipdnnCreateConvolutionDescriptor(&fc2_desc));

    CUDNN_CHECK(hipdnnCreateActivationDescriptor(&relu_activation));
    CUDNN_CHECK(hipdnnCreatePoolingDescriptor(&pooling_desc));

    // Now load weights and initialize layers
    loadWeights(weights_path);
    initializeLayers();
    checkTensorCoreUsage();
}

hipdnnConvolutionFwdAlgo_t TensorCNNInference::findBestConvAlgorithm(
    hipdnnTensorDescriptor_t input_desc,
    hipdnnFilterDescriptor_t filter_desc,
    hipdnnConvolutionDescriptor_t conv_desc,
    hipdnnTensorDescriptor_t output_desc,
    size_t* workspace_size) {
    
    const int requestedAlgoCount = 8;
    int returnedAlgoCount;
    std::vector<hipdnnConvolutionFwdAlgoPerf_t> perfResults(requestedAlgoCount);
    
    CUDNN_CHECK(cudnnGetConvolutionForwardAlgorithm_v7(cudnn,
        input_desc,
        filter_desc,
        conv_desc,
        output_desc,
        requestedAlgoCount,
        &returnedAlgoCount,
        perfResults.data()));

    // Find the fastest algorithm that uses Tensor Cores
    hipdnnConvolutionFwdAlgo_t bestAlgo = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM;
    float bestTime = std::numeric_limits<float>::max();
    
    for (int i = 0; i < returnedAlgoCount; i++) {
        if (perfResults[i].status == HIPDNN_STATUS_SUCCESS &&
            perfResults[i].mathType == HIPDNN_TENSOR_OP_MATH &&  // Require Tensor Core operations
            perfResults[i].time < bestTime) {
            bestTime = perfResults[i].time;
            bestAlgo = perfResults[i].algo;
            *workspace_size = std::max(*workspace_size, perfResults[i].memory);
        }
    }

    // If no Tensor Core algorithm was found, try again with any algorithm
    if (bestTime == std::numeric_limits<float>::max()) {
        for (int i = 0; i < returnedAlgoCount; i++) {
            if (perfResults[i].status == HIPDNN_STATUS_SUCCESS &&
                perfResults[i].time < bestTime) {
                bestTime = perfResults[i].time;
                bestAlgo = perfResults[i].algo;
                *workspace_size = std::max(*workspace_size, perfResults[i].memory);
            }
        }
    }

    return bestAlgo;
}

void TensorCNNInference::initializeLayers() {
    // Input dimensions configuration
    input_dims = {batch_size, 3, 32, 32};
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(input_desc, 
        HIPDNN_TENSOR_NCHW, 
        HIPDNN_DATA_HALF,  // Changed to HALF
        input_dims.n, input_dims.c, input_dims.h, input_dims.w));

    // Conv1 layer configuration
    CUDNN_CHECK(hipdnnSetFilter4dDescriptor(conv1_filter_desc, 
        HIPDNN_DATA_HALF, 
        HIPDNN_TENSOR_NCHW, 
        32, 3, 3, 3));
    
    CUDNN_CHECK(hipdnnSetConvolution2dDescriptor(conv1_desc, 
        1, 1,    // padding
        1, 1,    // stride
        1, 1,    // dilation
        HIPDNN_CROSS_CORRELATION, 
        HIPDNN_DATA_HALF));  // Changed to HALF
    
    CUDNN_CHECK(hipdnnSetConvolutionMathType(conv1_desc, HIPDNN_TENSOR_OP_MATH));
    
    // Get Conv1 output dimensions
    CUDNN_CHECK(hipdnnGetConvolution2dForwardOutputDim(conv1_desc, input_desc, 
        conv1_filter_desc, &conv1_dims.n, &conv1_dims.c, &conv1_dims.h, &conv1_dims.w));

    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(conv1_output_desc, 
        HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_HALF,  // Changed to HALF
        conv1_dims.n, conv1_dims.c, conv1_dims.h, conv1_dims.w));
    
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(conv1_bias_desc, 
        HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_HALF,  // Changed to HALF
        1, conv1_dims.c, 1, 1));

    // Pooling configuration
    CUDNN_CHECK(hipdnnSetPooling2dDescriptor(pooling_desc, 
        HIPDNN_POOLING_MAX,
        HIPDNN_NOT_PROPAGATE_NAN,
        2, 2,    // window size
        0, 0,    // padding
        2, 2));  // stride

    // Get Pool1 dimensions
    CUDNN_CHECK(hipdnnGetPooling2dForwardOutputDim(pooling_desc,
        conv1_output_desc,
        &pool1_dims.n, &pool1_dims.c, &pool1_dims.h, &pool1_dims.w));

    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(pool1_output_desc, 
        HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_HALF,  // Changed to HALF
        pool1_dims.n, pool1_dims.c, pool1_dims.h, pool1_dims.w));

    // Conv2 layer configuration
    CUDNN_CHECK(hipdnnSetFilter4dDescriptor(conv2_filter_desc, 
        HIPDNN_DATA_HALF, 
        HIPDNN_TENSOR_NCHW, 
        64, 32, 3, 3));
    
    CUDNN_CHECK(hipdnnSetConvolution2dDescriptor(conv2_desc, 
        1, 1,    // padding
        1, 1,    // stride
        1, 1,    // dilation
        HIPDNN_CROSS_CORRELATION, 
        HIPDNN_DATA_HALF));  // Changed to HALF
    
    CUDNN_CHECK(hipdnnSetConvolutionMathType(conv2_desc, HIPDNN_TENSOR_OP_MATH));
    
    // Get Conv2 output dimensions
    CUDNN_CHECK(hipdnnGetConvolution2dForwardOutputDim(conv2_desc, pool1_output_desc, 
        conv2_filter_desc, &conv2_dims.n, &conv2_dims.c, &conv2_dims.h, &conv2_dims.w));

    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(conv2_output_desc, 
        HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_HALF,  // Changed to HALF
        conv2_dims.n, conv2_dims.c, conv2_dims.h, conv2_dims.w));
    
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(conv2_bias_desc, 
        HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_HALF,  // Changed to HALF
        1, conv2_dims.c, 1, 1));

    // Pool2 layer configuration
    CUDNN_CHECK(hipdnnGetPooling2dForwardOutputDim(pooling_desc,
        conv2_output_desc,
        &pool2_dims.n, &pool2_dims.c, &pool2_dims.h, &pool2_dims.w));

    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(pool2_output_desc, 
        HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_HALF,  // Changed to HALF
        pool2_dims.n, pool2_dims.c, pool2_dims.h, pool2_dims.w));

    // Flatten pool2 output for FC layers
    int fc_input_size = pool2_dims.c * pool2_dims.h * pool2_dims.w;
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(pool2_flat_desc, 
        HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_HALF,  // Changed to HALF
        batch_size, fc_input_size, 1, 1));

    // FC1 layer configuration
    CUDNN_CHECK(hipdnnSetFilter4dDescriptor(fc1_filter_desc, 
        HIPDNN_DATA_HALF,
        HIPDNN_TENSOR_NCHW, 
        128, fc_input_size, 1, 1));
    
    CUDNN_CHECK(hipdnnSetConvolution2dDescriptor(fc1_desc,
        0, 0,    // padding
        1, 1,    // stride
        1, 1,    // dilation
        HIPDNN_CROSS_CORRELATION,
        HIPDNN_DATA_HALF));  // Changed to HALF
    
    CUDNN_CHECK(hipdnnSetConvolutionMathType(fc1_desc, HIPDNN_TENSOR_OP_MATH));
    
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(fc1_output_desc, 
        HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_HALF,  // Changed to HALF
        batch_size, 128, 1, 1));
    
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(fc1_bias_desc, 
        HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_HALF,  // Changed to HALF
        1, 128, 1, 1));

    // FC2 layer configuration
    CUDNN_CHECK(hipdnnSetFilter4dDescriptor(fc2_filter_desc, 
        HIPDNN_DATA_HALF,
        HIPDNN_TENSOR_NCHW, 
        10, 128, 1, 1));
    
    CUDNN_CHECK(hipdnnSetConvolution2dDescriptor(fc2_desc,
        0, 0,    // padding
        1, 1,    // stride
        1, 1,    // dilation
        HIPDNN_CROSS_CORRELATION,
        HIPDNN_DATA_HALF));  // Changed to HALF
    
    CUDNN_CHECK(hipdnnSetConvolutionMathType(fc2_desc, HIPDNN_TENSOR_OP_MATH));
    
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(fc2_output_desc, 
        HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_HALF,  // Changed to HALF
        batch_size, 10, 1, 1));
    
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(fc2_bias_desc, 
        HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_HALF,  // Changed to HALF
        1, 10, 1, 1));

    // ReLU activation configuration
    CUDNN_CHECK(hipdnnSetActivationDescriptor(relu_activation,
        HIPDNN_ACTIVATION_RELU,
        HIPDNN_NOT_PROPAGATE_NAN,
        0.0));

    // Find best algorithms for convolution operations
    workspace_size = 0;
    conv1_algo = findBestConvAlgorithm(input_desc, conv1_filter_desc, conv1_desc, 
                                     conv1_output_desc, &workspace_size);
    conv2_algo = findBestConvAlgorithm(pool1_output_desc, conv2_filter_desc, conv2_desc, 
                                     conv2_output_desc, &workspace_size);
    fc1_algo = findBestConvAlgorithm(pool2_flat_desc, fc1_filter_desc, fc1_desc, 
                                   fc1_output_desc, &workspace_size);
    fc2_algo = findBestConvAlgorithm(fc1_output_desc, fc2_filter_desc, fc2_desc, 
                                   fc2_output_desc, &workspace_size);

    // Allocate workspace memory
    CUDA_CHECK(hipMalloc(&d_workspace, workspace_size));

    // Allocate device memory for intermediate results
    CUDA_CHECK(hipMalloc(&d_input, batch_size * 3 * 32 * 32 * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_conv1_output, batch_size * conv1_dims.c * conv1_dims.h * conv1_dims.w * sizeof(half)));
    CUDA_CHECK(hipMalloc(&d_pool1_output, batch_size * pool1_dims.c * pool1_dims.h * pool1_dims.w * sizeof(half)));
    CUDA_CHECK(hipMalloc(&d_conv2_output, batch_size * conv2_dims.c * conv2_dims.h * conv2_dims.w * sizeof(half)));
    CUDA_CHECK(hipMalloc(&d_pool2_output, batch_size * pool2_dims.c * pool2_dims.h * pool2_dims.w * sizeof(half)));
    CUDA_CHECK(hipMalloc(&d_fc1_output, batch_size * 128 * sizeof(half)));
    CUDA_CHECK(hipMalloc(&d_fc2_output, batch_size * 10 * sizeof(half)));

    std::cout << "Layer initialization complete." << std::endl;
    std::cout << "Workspace size: " << workspace_size / (1024.0 * 1024.0) << " MB" << std::endl;
}

void TensorCNNInference::checkTensorCoreUsage() {
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, 0));
    
    std::cout << "\nGPU Configuration:" << std::endl;
    std::cout << "GPU: " << prop.name << std::endl;
    std::cout << "Compute Capability: " << prop.major << "." << prop.minor << std::endl;
    
    // Tensor Cores are available on:
    // - Volta (7.0) and above for FP16
    // - Ampere (8.0) and above for TF32
    bool hasTensorCores = false;
    bool supportsTF32 = false;
    
    if (prop.major >= 7) {
        hasTensorCores = true;
        if (prop.major >= 8) {
            supportsTF32 = true;
        }
    }
    
    std::cout << "Tensor Cores Available: " << (hasTensorCores ? "Yes" : "No") << std::endl;
    if (hasTensorCores) {
        std::cout << "TF32 Support: " << (supportsTF32 ? "Yes" : "No") << std::endl;
    }
    
    // Check math type configuration for each convolution
    hipdnnMathType_t mathType;
    CUDNN_CHECK(cudnnGetConvolutionMathType(conv1_desc, &mathType));
    std::cout << "\nConvolution Layer Math Types:" << std::endl;
    std::cout << "Conv1: " << 
        (mathType == HIPDNN_TENSOR_OP_MATH ? "Tensor Core" : "Standard") << std::endl;
    
    CUDNN_CHECK(cudnnGetConvolutionMathType(conv2_desc, &mathType));
    std::cout << "Conv2: " << 
        (mathType == HIPDNN_TENSOR_OP_MATH ? "Tensor Core" : "Standard") << std::endl;
    
    CUDNN_CHECK(cudnnGetConvolutionMathType(fc1_desc, &mathType));
    std::cout << "FC1: " << 
        (mathType == HIPDNN_TENSOR_OP_MATH ? "Tensor Core" : "Standard") << std::endl;
    
    CUDNN_CHECK(cudnnGetConvolutionMathType(fc2_desc, &mathType));
    std::cout << "FC2: " << 
        (mathType == HIPDNN_TENSOR_OP_MATH ? "Tensor Core" : "Standard") << std::endl;
    
    std::cout << "\nWorkspace Size: " << workspace_size / (1024.0 * 1024.0) << " MB" << std::endl;
}

void TensorCNNInference::infer(const std::vector<float>& input_data) {
    // Use float values for alpha/beta
    const float alpha_f = 1.0f;
    const float beta_f = 0.0f;
    const void* alpha_ptr = &alpha_f;
    const void* beta_ptr = &beta_f;

    // Verify input data size and copy to device
    size_t expected_input_size = batch_size * 3 * 32 * 32;
    if (input_data.size() != expected_input_size) {
        throw std::runtime_error("Input data size mismatch");
    }

    // Copy input FP32 data to device
    CUDA_CHECK(hipMemcpy(d_input, input_data.data(), 
                         input_data.size() * sizeof(float), 
                         hipMemcpyHostToDevice));

    // Convert input from FP32 to FP16
    half* d_input_half;
    CUDA_CHECK(hipMalloc(&d_input_half, input_data.size() * sizeof(half)));
    convertToFP16((float*)d_input, d_input_half, input_data.size());

    // Conv1 + ReLU
    CUDNN_CHECK(hipdnnConvolutionForward(cudnn,
        alpha_ptr,
        input_desc, d_input_half,
        conv1_filter_desc, d_conv1_weight,
        conv1_desc, conv1_algo,
        d_workspace, workspace_size,
        beta_ptr,
        conv1_output_desc, d_conv1_output));

    // Add bias
    CUDNN_CHECK(hipdnnAddTensor(cudnn,
        alpha_ptr,
        conv1_bias_desc, d_conv1_bias,
        alpha_ptr,
        conv1_output_desc, d_conv1_output));

    // ReLU activation
    CUDNN_CHECK(hipdnnActivationForward(cudnn, relu_activation,
        alpha_ptr,
        conv1_output_desc, d_conv1_output,
        beta_ptr,
        conv1_output_desc, d_conv1_output));

    // MaxPool1
    CUDNN_CHECK(hipdnnPoolingForward(cudnn, pooling_desc,
        alpha_ptr,
        conv1_output_desc, d_conv1_output,
        beta_ptr,
        pool1_output_desc, d_pool1_output));

    // Conv2 + ReLU
    CUDNN_CHECK(hipdnnConvolutionForward(cudnn,
        alpha_ptr,
        pool1_output_desc, d_pool1_output,
        conv2_filter_desc, d_conv2_weight,
        conv2_desc, conv2_algo,
        d_workspace, workspace_size,
        beta_ptr,
        conv2_output_desc, d_conv2_output));

    // Add bias
    CUDNN_CHECK(hipdnnAddTensor(cudnn,
        alpha_ptr,
        conv2_bias_desc, d_conv2_bias,
        alpha_ptr,
        conv2_output_desc, d_conv2_output));

    // ReLU activation
    CUDNN_CHECK(hipdnnActivationForward(cudnn, relu_activation,
        alpha_ptr,
        conv2_output_desc, d_conv2_output,
        beta_ptr,
        conv2_output_desc, d_conv2_output));

    // MaxPool2
    CUDNN_CHECK(hipdnnPoolingForward(cudnn, pooling_desc,
        alpha_ptr,
        conv2_output_desc, d_conv2_output,
        beta_ptr,
        pool2_output_desc, d_pool2_output));

    // FC1 + ReLU
    CUDNN_CHECK(hipdnnConvolutionForward(cudnn,
        alpha_ptr,
        pool2_flat_desc, d_pool2_output,
        fc1_filter_desc, d_fc1_weight,
        fc1_desc, fc1_algo,
        d_workspace, workspace_size,
        beta_ptr,
        fc1_output_desc, d_fc1_output));

    // Add bias
    CUDNN_CHECK(hipdnnAddTensor(cudnn,
        alpha_ptr,
        fc1_bias_desc, d_fc1_bias,
        alpha_ptr,
        fc1_output_desc, d_fc1_output));

    // ReLU activation
    CUDNN_CHECK(hipdnnActivationForward(cudnn, relu_activation,
        alpha_ptr,
        fc1_output_desc, d_fc1_output,
        beta_ptr,
        fc1_output_desc, d_fc1_output));

    // FC2 (final layer)
    CUDNN_CHECK(hipdnnConvolutionForward(cudnn,
        alpha_ptr,
        fc1_output_desc, d_fc1_output,
        fc2_filter_desc, d_fc2_weight,
        fc2_desc, fc2_algo,
        d_workspace, workspace_size,
        beta_ptr,
        fc2_output_desc, d_fc2_output));

    // Add final bias
    CUDNN_CHECK(hipdnnAddTensor(cudnn,
        alpha_ptr,
        fc2_bias_desc, d_fc2_bias,
        alpha_ptr,
        fc2_output_desc, d_fc2_output));

    // Clean up temporary FP16 input buffer
    CUDA_CHECK(hipFree(d_input_half));

    // Check for any CUDA errors
    CUDA_CHECK(hipGetLastError());
}

std::vector<float> TensorCNNInference::getOutput() {
    size_t output_size = batch_size * 10;
    std::vector<float> output(output_size);

    // Convert FP16 output to FP32
    float* d_output_float;
    CUDA_CHECK(hipMalloc(&d_output_float, output_size * sizeof(float)));
    convertToFP32((half*)d_fc2_output, d_output_float, output_size);

    // Copy the FP32 output from device to host
    CUDA_CHECK(hipMemcpy(output.data(), d_output_float, 
                         output.size() * sizeof(float), hipMemcpyDeviceToHost));
    
    hipFree(d_output_float);

    // Apply softmax normalization for each sample
    for (int i = 0; i < batch_size; ++i) {
        float max_val = *std::max_element(output.begin() + i * 10, output.begin() + (i + 1) * 10);
        float sum = 0.0f;

        for (int j = 0; j < 10; ++j) {
            output[i * 10 + j] = std::exp(output[i * 10 + j] - max_val);
            sum += output[i * 10 + j];
        }

        for (int j = 0; j < 10; ++j) {
            output[i * 10 + j] /= sum;
        }
    }

    return output;
}

TensorCNNInference::~TensorCNNInference() {
    // Free device memory
    hipFree(d_input);
    hipFree(d_conv1_weight);
    hipFree(d_conv1_bias);
    hipFree(d_conv1_output);
    hipFree(d_pool1_output);
    hipFree(d_conv2_weight);
    hipFree(d_conv2_bias);
    hipFree(d_conv2_output);
    hipFree(d_pool2_output);
    hipFree(d_fc1_weight);
    hipFree(d_fc1_bias);
    hipFree(d_fc1_output);
    hipFree(d_fc2_weight);
    hipFree(d_fc2_bias);
    hipFree(d_fc2_output);
    hipFree(d_workspace);

    // Destroy descriptors
    hipdnnDestroyTensorDescriptor(input_desc);
    hipdnnDestroyTensorDescriptor(conv1_output_desc);
    hipdnnDestroyTensorDescriptor(pool1_output_desc);
    hipdnnDestroyTensorDescriptor(conv2_output_desc);
    hipdnnDestroyTensorDescriptor(pool2_output_desc);
    hipdnnDestroyTensorDescriptor(pool2_flat_desc);
    hipdnnDestroyTensorDescriptor(fc1_output_desc);
    hipdnnDestroyTensorDescriptor(fc2_output_desc);
    
    hipdnnDestroyFilterDescriptor(conv1_filter_desc);
    hipdnnDestroyFilterDescriptor(conv2_filter_desc);
    hipdnnDestroyFilterDescriptor(fc1_filter_desc);
    hipdnnDestroyFilterDescriptor(fc2_filter_desc);
    
    hipdnnDestroyTensorDescriptor(conv1_bias_desc);
    hipdnnDestroyTensorDescriptor(conv2_bias_desc);
    hipdnnDestroyTensorDescriptor(fc1_bias_desc);
    hipdnnDestroyTensorDescriptor(fc2_bias_desc);
    
    hipdnnDestroyConvolutionDescriptor(conv1_desc);
    hipdnnDestroyConvolutionDescriptor(conv2_desc);
    hipdnnDestroyConvolutionDescriptor(fc1_desc);
    hipdnnDestroyConvolutionDescriptor(fc2_desc);
    
    hipdnnDestroyActivationDescriptor(relu_activation);
    hipdnnDestroyPoolingDescriptor(pooling_desc);
    
    hipdnnDestroy(cudnn);
}

void TensorCNNInference::loadWeights(const std::string& weights_path) {
    std::cout << "Loading FP16 model weights..." << std::endl;

    // Build full paths to weight files
    std::string conv1_weight_path = weights_path + "/conv1.weight_fp16.bin";
    std::string conv1_bias_path = weights_path + "/conv1.bias_fp16.bin";
    std::string conv2_weight_path = weights_path + "/conv2.weight_fp16.bin";
    std::string conv2_bias_path = weights_path + "/conv2.bias_fp16.bin";
    std::string fc1_weight_path = weights_path + "/fc1.weight_fp16.bin";
    std::string fc1_bias_path = weights_path + "/fc1.bias_fp16.bin";
    std::string fc2_weight_path = weights_path + "/fc2.weight_fp16.bin";
    std::string fc2_bias_path = weights_path + "/fc2.bias_fp16.bin";

    // Load the FP16 weights from binary files
    auto conv1_weights = loadBinaryFile<half>(conv1_weight_path);
    auto conv1_biases = loadBinaryFile<half>(conv1_bias_path);
    auto conv2_weights = loadBinaryFile<half>(conv2_weight_path);
    auto conv2_biases = loadBinaryFile<half>(conv2_bias_path);
    auto fc1_weights = loadBinaryFile<half>(fc1_weight_path);
    auto fc1_biases = loadBinaryFile<half>(fc1_bias_path);
    auto fc2_weights = loadBinaryFile<half>(fc2_weight_path);
    auto fc2_biases = loadBinaryFile<half>(fc2_bias_path);

    // Verify sizes based on PyTorch model shapes
    const size_t conv1_weights_size = 32 * 3 * 3 * 3;      // (32, 3, 3, 3)
    const size_t conv1_bias_size = 32;                     // (32,)
    const size_t conv2_weights_size = 64 * 32 * 3 * 3;     // (64, 32, 3, 3)
    const size_t conv2_bias_size = 64;                     // (64,)
    const size_t fc1_weights_size = 128 * (64 * 8 * 8);    // (128, 4096)
    const size_t fc1_bias_size = 128;                      // (128,)
    const size_t fc2_weights_size = 10 * 128;              // (10, 128)
    const size_t fc2_bias_size = 10;                       // (10,)

    // Verify sizes
    if (conv1_weights.size() != conv1_weights_size ||
        conv1_biases.size() != conv1_bias_size ||
        conv2_weights.size() != conv2_weights_size ||
        conv2_biases.size() != conv2_bias_size ||
        fc1_weights.size() != fc1_weights_size ||
        fc1_biases.size() != fc1_bias_size ||
        fc2_weights.size() != fc2_weights_size ||
        fc2_biases.size() != fc2_bias_size) {
        
        std::stringstream error_msg;
        error_msg << "Weight file sizes do not match expected dimensions.\n";
        throw std::runtime_error(error_msg.str());
    }    
    // Allocate and copy weights to device
    CUDA_CHECK(hipMalloc(&d_conv1_weight, conv1_weights_size * sizeof(half)));
    CUDA_CHECK(hipMalloc(&d_conv1_bias, conv1_bias_size * sizeof(half)));
    CUDA_CHECK(hipMemcpy(d_conv1_weight, conv1_weights.data(), 
                         conv1_weights_size * sizeof(half), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_conv1_bias, conv1_biases.data(), 
                         conv1_bias_size * sizeof(half), hipMemcpyHostToDevice));
    
    CUDA_CHECK(hipMalloc(&d_conv2_weight, conv2_weights_size * sizeof(half)));
    CUDA_CHECK(hipMalloc(&d_conv2_bias, conv2_bias_size * sizeof(half)));
    CUDA_CHECK(hipMemcpy(d_conv2_weight, conv2_weights.data(),
                         conv2_weights_size * sizeof(half), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_conv2_bias, conv2_biases.data(),
                         conv2_bias_size * sizeof(half), hipMemcpyHostToDevice));

    CUDA_CHECK(hipMalloc(&d_fc1_weight, fc1_weights_size * sizeof(half)));
    CUDA_CHECK(hipMalloc(&d_fc1_bias, fc1_bias_size * sizeof(half)));
    CUDA_CHECK(hipMemcpy(d_fc1_weight, fc1_weights.data(),
                         fc1_weights_size * sizeof(half), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_fc1_bias, fc1_biases.data(),
                         fc1_bias_size * sizeof(half), hipMemcpyHostToDevice));

    CUDA_CHECK(hipMalloc(&d_fc2_weight, fc2_weights_size * sizeof(half)));
    CUDA_CHECK(hipMalloc(&d_fc2_bias, fc2_bias_size * sizeof(half)));
    CUDA_CHECK(hipMemcpy(d_fc2_weight, fc2_weights.data(),
                         fc2_weights_size * sizeof(half), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_fc2_bias, fc2_biases.data(),
                         fc2_bias_size * sizeof(half), hipMemcpyHostToDevice));

    // Verify weights were loaded successfully
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        throw std::runtime_error(std::string("CUDA error while loading weights: ") +
                               hipGetErrorString(error));
    }

    std::cout << "Successfully loaded all FP16 weights to GPU." << std::endl;
}

void parseArguments(int argc, char** argv, int& gpu_id, int& repeat_factor, int& batch_size, std::string& data_path, std::string& weights_path) {
    if (argc >= 6) {
        gpu_id = std::atoi(argv[1]);
        repeat_factor = std::atoi(argv[2]);
        batch_size = std::atoi(argv[3]);
        data_path = argv[4];
        weights_path = argv[5];
    } else {
        std::cerr << "Usage: " << argv[0] << " <gpu_id> <repeat_factor> <batch_size> <data_path> <weights_path>" << std::endl;
        std::cerr << "Example: " << argv[0] << " 0 10 256 /path/to/data/validation /path/to/data/weights" << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

int main(int argc, char** argv) {
    int gpu_id = 0;
    int repeat_factor = 1;
    int batch_size = 256;
    std::string data_path;
    std::string weights_path;

    parseArguments(argc, argv, gpu_id, repeat_factor, batch_size, data_path, weights_path);
    CUDA_CHECK(hipSetDevice(gpu_id));

    std::cout << "Running on GPU: " << gpu_id << std::endl;
    std::cout << "Repeat factor: " << repeat_factor << std::endl;
    std::cout << "Batch size: " << batch_size << std::endl;

    try {
        auto validation_images_path = data_path + "/validation_images.bin";
        auto validation_labels_path = data_path + "/validation_labels.bin";

        auto validation_images = loadBinaryFile<float>(validation_images_path);
        auto validation_labels = loadBinaryFile<int>(validation_labels_path);

        size_t image_size = 3 * 32 * 32;
        std::vector<std::vector<float>> images;
        for (size_t i = 0; i < validation_images.size(); i += image_size) {
            images.push_back(std::vector<float>(validation_images.begin() + i, 
                                              validation_images.begin() + i + image_size));
        }

        std::vector<std::vector<float>> repeated_images;
        std::vector<int> repeated_labels;

        for (int i = 0; i < repeat_factor; ++i) {
            repeated_images.insert(repeated_images.end(), images.begin(), images.end());
            repeated_labels.insert(repeated_labels.end(), validation_labels.begin(), validation_labels.end());
        }
        
        size_t total_images = repeated_images.size();
        std::cout << "Total images after repeating: " << total_images << std::endl;

        TensorCNNInference cnn(batch_size, weights_path);
        cnn.checkTensorCoreUsage();

        hipEvent_t start, stop;
        CUDA_CHECK(hipEventCreate(&start));
        CUDA_CHECK(hipEventCreate(&stop));

        size_t correct_count = 0;
        float total_time = 0.0f;

        // Warmup with first batch
        std::vector<float> warmup_batch;
        warmup_batch.reserve(batch_size * image_size);
        for (int i = 0; i < batch_size && i < total_images; ++i) {
            warmup_batch.insert(warmup_batch.end(), repeated_images[i].begin(), repeated_images[i].end());
        }
        for (int i = 0; i < 10; i++) {
            cnn.infer(warmup_batch);
        }

        // Main evaluation loop with batching
        size_t total_batches = (total_images + batch_size - 1) / batch_size;
        std::cout << "Starting main evaluation..." << std::endl;

        for (size_t batch_idx = 0; batch_idx < total_batches; ++batch_idx) {
            size_t batch_start = batch_idx * batch_size;
            size_t batch_end = std::min(batch_start + batch_size, total_images);
            size_t current_batch_size = batch_end - batch_start;

            std::vector<float> batch_input;
            batch_input.reserve(batch_size * image_size);

            // Load actual images
            for (size_t i = batch_start; i < batch_end; ++i) {
                batch_input.insert(batch_input.end(), repeated_images[i].begin(), repeated_images[i].end());
            }

            // Pad the batch if necessary
            if (current_batch_size < batch_size) {
                // Duplicate the last image to fill the batch
                const auto& last_image = repeated_images[batch_end - 1];
                for (size_t i = current_batch_size; i < batch_size; ++i) {
                    batch_input.insert(batch_input.end(), last_image.begin(), last_image.end());
                }
            }

            CUDA_CHECK(hipEventRecord(start));
            cnn.infer(batch_input);
            std::vector<float> output = cnn.getOutput();
            CUDA_CHECK(hipEventRecord(stop));
            CUDA_CHECK(hipEventSynchronize(stop));
            
            float milliseconds = 0;
            CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
            total_time += milliseconds;

            for (size_t i = 0; i < current_batch_size; ++i) {
                int predicted_label = std::distance(
                    output.begin() + i * 10,
                    std::max_element(output.begin() + i * 10, output.begin() + (i + 1) * 10)
                );
                if (predicted_label == repeated_labels[batch_start + i]) {
                    ++correct_count;
                }
            }

            if (batch_idx % 10 == 0) {
                float running_accuracy = (static_cast<float>(correct_count) / ((batch_idx + 1) * batch_size)) * 100.0f;
                std::cout << "Processed " << (batch_idx + 1) * batch_size << "/" << total_images 
                         << " images. Accuracy: " << std::fixed 
                         << std::setprecision(2) << running_accuracy << "%" << std::endl;
            }
        }

        float accuracy = static_cast<float>(correct_count) / total_images * 100.0f;
        float avg_time = total_time / total_batches;
        float throughput = (batch_size * 1000.0f) / avg_time;

        std::cout << "\n=== Final Results ===" << std::endl;
        std::cout << "Model type: Tensor Core FP16" << std::endl;
        std::cout << "Batch size: " << batch_size << std::endl;
        std::cout << "Total images: " << total_images << std::endl;
        std::cout << "Correct predictions: " << correct_count << std::endl;
        std::cout << "Accuracy: " << std::fixed << std::setprecision(2) << accuracy << "%" << std::endl;
        std::cout << "Average batch time: " << std::fixed << std::setprecision(3) << avg_time << " ms" << std::endl;
        std::cout << "Throughput: " << std::fixed << std::setprecision(1) << throughput << " images/sec" << std::endl;
        std::cout << "Total time: " << std::fixed << std::setprecision(2) << total_time / 1000.0f << " seconds" << std::endl;

        CUDA_CHECK(hipEventDestroy(start));
        CUDA_CHECK(hipEventDestroy(stop));

    } catch (const std::exception& e) {
        std::cerr << "Fatal error: " << e.what() << std::endl;
        return 1;
    }

    return 0;
}
