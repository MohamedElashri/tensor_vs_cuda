#include "hip/hip_runtime.h"
// CUDA Cores + Tensor Cores Inference Engine

#include </usr/include/hipDNN.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <cassert>
#include <algorithm>
#include <chrono>
#include <iomanip>

// Load binary data from file
template <typename T>
std::vector<T> loadBinaryFile(const std::string& filename) {
    std::ifstream file(filename, std::ios::binary);
    if (!file.is_open()) {
        std::cerr << "Error: Could not open file " << filename << std::endl;
        std::exit(EXIT_FAILURE);
    }
    file.seekg(0, std::ios::end);
    size_t size = file.tellg() / sizeof(T);
    file.seekg(0, std::ios::beg);
    std::vector<T> buffer(size);
    file.read(reinterpret_cast<char*>(buffer.data()), size * sizeof(T));
    file.close();
    return buffer;
}

// Helper function to handle CuDNN errors
#define CUDNN_CHECK(call) {                                                        \
    hipdnnStatus_t err = call;                                                     \
    if (err != HIPDNN_STATUS_SUCCESS) {                                           \
        std::cerr << "CuDNN Error at " << __FILE__ << ":" << __LINE__ << ": "   \
                  << hipdnnGetErrorString(err) << std::endl;                      \
        std::exit(EXIT_FAILURE);                                                 \
    }                                                                            \
}

// Helper function to handle CUDA errors
#define CUDA_CHECK(call) {                                                        \
    hipError_t err = call;                                                      \
    if (err != hipSuccess) {                                                    \
        std::cerr << "CUDA Error at " << __FILE__ << ":" << __LINE__ << ": "    \
                  << hipGetErrorString(err) << std::endl;                       \
        std::exit(EXIT_FAILURE);                                                 \
    }                                                                            \
}

class TensorCoreCNNInference {
public:
    TensorCoreCNNInference();
    ~TensorCoreCNNInference();
    void loadWeights();
    void initializeLayers();
    void checkTensorCoreUsage();
    void infer(const std::vector<float>& input_data);
    std::vector<float> getOutput();
    void evaluate(const std::vector<std::vector<float>>& images, 
                 const std::vector<int>& labels);

private:
    hipdnnHandle_t cudnn;
    
    // Layer descriptors
    hipdnnTensorDescriptor_t input_desc;
    hipdnnTensorDescriptor_t conv1_output_desc;
    hipdnnTensorDescriptor_t pool1_output_desc;
    hipdnnTensorDescriptor_t conv2_output_desc;
    hipdnnTensorDescriptor_t pool2_output_desc;
    hipdnnTensorDescriptor_t pool2_flat_desc;
    hipdnnTensorDescriptor_t fc1_output_desc;
    hipdnnTensorDescriptor_t fc2_output_desc;

    // Filter descriptors
    hipdnnFilterDescriptor_t conv1_filter_desc;
    hipdnnFilterDescriptor_t conv2_filter_desc;
    hipdnnFilterDescriptor_t fc1_filter_desc;
    hipdnnFilterDescriptor_t fc2_filter_desc;
    
    // Bias descriptors
    hipdnnTensorDescriptor_t conv1_bias_desc;
    hipdnnTensorDescriptor_t conv2_bias_desc;
    hipdnnTensorDescriptor_t fc1_bias_desc;
    hipdnnTensorDescriptor_t fc2_bias_desc;
    
    // Convolution descriptors
    hipdnnConvolutionDescriptor_t conv1_desc;
    hipdnnConvolutionDescriptor_t conv2_desc;
    hipdnnConvolutionDescriptor_t fc1_desc;
    hipdnnConvolutionDescriptor_t fc2_desc;
    
    // Activation and pooling descriptors
    hipdnnActivationDescriptor_t relu_activation;
    hipdnnPoolingDescriptor_t pooling_desc;

    // Convolution algorithms
    hipdnnConvolutionFwdAlgo_t conv1_algo;
    hipdnnConvolutionFwdAlgo_t conv2_algo;
    hipdnnConvolutionFwdAlgo_t fc1_algo;
    hipdnnConvolutionFwdAlgo_t fc2_algo;

    // Device memory pointers
    float *d_input;
    float *d_conv1_weight, *d_conv1_bias, *d_conv1_output;
    float *d_pool1_output;
    float *d_conv2_weight, *d_conv2_bias, *d_conv2_output;
    float *d_pool2_output;
    float *d_fc1_weight, *d_fc1_bias, *d_fc1_output;
    float *d_fc2_weight, *d_fc2_bias, *d_fc2_output;

    // Workspace for convolutions
    size_t workspace_size;
    void *d_workspace;

    // Dimensions
    int batch_size;
    struct LayerDims {
        int n, c, h, w;
    };
    LayerDims input_dims, conv1_dims, pool1_dims, conv2_dims, pool2_dims, 
              fc1_dims, fc2_dims;

    // Find best convolution algorithm that uses Tensor Cores
    hipdnnConvolutionFwdAlgo_t findBestConvAlgorithm(
        hipdnnTensorDescriptor_t input_desc,
        hipdnnFilterDescriptor_t filter_desc,
        hipdnnConvolutionDescriptor_t conv_desc,
        hipdnnTensorDescriptor_t output_desc,
        size_t* workspace_size);
};

TensorCoreCNNInference::TensorCoreCNNInference() : batch_size(1) {
    std::cout << "Initializing TensorCore CNN..." << std::endl;
    
    // Create cuDNN handle
    CUDNN_CHECK(hipdnnCreate(&cudnn));
    
    // Create descriptors
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&input_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&conv1_output_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&pool1_output_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&conv2_output_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&pool2_output_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&pool2_flat_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&fc1_output_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&fc2_output_desc));

    CUDNN_CHECK(hipdnnCreateFilterDescriptor(&conv1_filter_desc));
    CUDNN_CHECK(hipdnnCreateFilterDescriptor(&conv2_filter_desc));
    CUDNN_CHECK(hipdnnCreateFilterDescriptor(&fc1_filter_desc));
    CUDNN_CHECK(hipdnnCreateFilterDescriptor(&fc2_filter_desc));

    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&conv1_bias_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&conv2_bias_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&fc1_bias_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&fc2_bias_desc));

    CUDNN_CHECK(hipdnnCreateConvolutionDescriptor(&conv1_desc));
    CUDNN_CHECK(hipdnnCreateConvolutionDescriptor(&conv2_desc));
    CUDNN_CHECK(hipdnnCreateConvolutionDescriptor(&fc1_desc));
    CUDNN_CHECK(hipdnnCreateConvolutionDescriptor(&fc2_desc));

    CUDNN_CHECK(hipdnnCreateActivationDescriptor(&relu_activation));
    CUDNN_CHECK(hipdnnCreatePoolingDescriptor(&pooling_desc));

    loadWeights();
    initializeLayers();
    checkTensorCoreUsage();
}

hipdnnConvolutionFwdAlgo_t TensorCoreCNNInference::findBestConvAlgorithm(
    hipdnnTensorDescriptor_t input_desc,
    hipdnnFilterDescriptor_t filter_desc,
    hipdnnConvolutionDescriptor_t conv_desc,
    hipdnnTensorDescriptor_t output_desc,
    size_t* workspace_size) {
    
    const int requestedAlgoCount = 8;
    int returnedAlgoCount;
    std::vector<hipdnnConvolutionFwdAlgoPerf_t> perfResults(requestedAlgoCount);
    
    CUDNN_CHECK(cudnnGetConvolutionForwardAlgorithm_v7(cudnn,
        input_desc,
        filter_desc,
        conv_desc,
        output_desc,
        requestedAlgoCount,
        &returnedAlgoCount,
        perfResults.data()));

    // Find the fastest algorithm that uses Tensor Cores
    hipdnnConvolutionFwdAlgo_t bestAlgo = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM;
    float bestTime = std::numeric_limits<float>::max();
    
    for (int i = 0; i < returnedAlgoCount; i++) {
        if (perfResults[i].status == HIPDNN_STATUS_SUCCESS &&
            perfResults[i].mathType == HIPDNN_TENSOR_OP_MATH &&
            perfResults[i].time < bestTime) {
            bestTime = perfResults[i].time;
            bestAlgo = perfResults[i].algo;
            *workspace_size = std::max(*workspace_size, perfResults[i].memory);
        }
    }

    return bestAlgo;
}



void TensorCoreCNNInference::initializeLayers() {
    // Input layer: 3x32x32
    input_dims = {batch_size, 3, 32, 32};
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(input_desc, HIPDNN_TENSOR_NCHW, 
        HIPDNN_DATA_FLOAT, input_dims.n, input_dims.c, input_dims.h, input_dims.w));

    // Conv1 layer setup
    conv1_dims = {batch_size, 32, 32, 32};  // Output size after padding
    CUDNN_CHECK(hipdnnSetFilter4dDescriptor(conv1_filter_desc, HIPDNN_DATA_FLOAT, 
        HIPDNN_TENSOR_NCHW, 32, 3, 3, 3));
    
    CUDNN_CHECK(hipdnnSetConvolution2dDescriptor(conv1_desc, 
        1, 1,    // padding
        1, 1,    // stride
        1, 1,    // dilation
        HIPDNN_CROSS_CORRELATION, 
        HIPDNN_DATA_FLOAT));
    
    // Enable Tensor Core operation for conv1
    CUDNN_CHECK(hipdnnSetConvolutionMathType(conv1_desc, HIPDNN_TENSOR_OP_MATH));
    
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(conv1_output_desc, HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_FLOAT, conv1_dims.n, conv1_dims.c, conv1_dims.h, conv1_dims.w));
    
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(conv1_bias_desc, HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_FLOAT, 1, conv1_dims.c, 1, 1));

    // Pooling setup
    CUDNN_CHECK(hipdnnSetPooling2dDescriptor(pooling_desc, 
        HIPDNN_POOLING_MAX,
        HIPDNN_NOT_PROPAGATE_NAN,
        2, 2,    // window height, width
        0, 0,    // padding height, width
        2, 2));  // stride height, width

    // Get Pool1 dimensions
    pool1_dims = {conv1_dims.n, conv1_dims.c, conv1_dims.h/2, conv1_dims.w/2};
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(pool1_output_desc, HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_FLOAT, pool1_dims.n, pool1_dims.c, pool1_dims.h, pool1_dims.w));

    // Conv2 setup
    conv2_dims = {pool1_dims.n, 64, pool1_dims.h, pool1_dims.w};
    CUDNN_CHECK(hipdnnSetFilter4dDescriptor(conv2_filter_desc, HIPDNN_DATA_FLOAT, 
        HIPDNN_TENSOR_NCHW, 64, 32, 3, 3));
    
    CUDNN_CHECK(hipdnnSetConvolution2dDescriptor(conv2_desc, 
        1, 1,    // padding
        1, 1,    // stride
        1, 1,    // dilation
        HIPDNN_CROSS_CORRELATION, 
        HIPDNN_DATA_FLOAT));
    
    // Enable Tensor Core operation for conv2
    CUDNN_CHECK(hipdnnSetConvolutionMathType(conv2_desc, HIPDNN_TENSOR_OP_MATH));
    
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(conv2_output_desc, HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_FLOAT, conv2_dims.n, conv2_dims.c, conv2_dims.h, conv2_dims.w));
    
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(conv2_bias_desc, HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_FLOAT, 1, conv2_dims.c, 1, 1));

    // Pool2 setup
    pool2_dims = {conv2_dims.n, conv2_dims.c, conv2_dims.h/2, conv2_dims.w/2};
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(pool2_output_desc, HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_FLOAT, pool2_dims.n, pool2_dims.c, pool2_dims.h, pool2_dims.w));

    // Setup flattened pool2 for FC layers
    int fc_input_size = pool2_dims.c * pool2_dims.h * pool2_dims.w;
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(pool2_flat_desc, HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_FLOAT, batch_size, fc_input_size, 1, 1));

    // FC1 setup
    fc1_dims = {batch_size, 128, 1, 1};
    CUDNN_CHECK(hipdnnSetFilter4dDescriptor(fc1_filter_desc, HIPDNN_DATA_FLOAT,
        HIPDNN_TENSOR_NCHW, 128, fc_input_size, 1, 1));
    
    CUDNN_CHECK(hipdnnSetConvolution2dDescriptor(fc1_desc,
        0, 0,    // padding
        1, 1,    // stride
        1, 1,    // dilation
        HIPDNN_CROSS_CORRELATION,
        HIPDNN_DATA_FLOAT));
    
    // Enable Tensor Core operation for FC1
    CUDNN_CHECK(hipdnnSetConvolutionMathType(fc1_desc, HIPDNN_TENSOR_OP_MATH));
    
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(fc1_output_desc, HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_FLOAT, fc1_dims.n, fc1_dims.c, fc1_dims.h, fc1_dims.w));
    
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(fc1_bias_desc, HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_FLOAT, 1, fc1_dims.c, 1, 1));

    // FC2 setup
    fc2_dims = {batch_size, 10, 1, 1};
    CUDNN_CHECK(hipdnnSetFilter4dDescriptor(fc2_filter_desc, HIPDNN_DATA_FLOAT,
        HIPDNN_TENSOR_NCHW, 10, 128, 1, 1));
    
    CUDNN_CHECK(hipdnnSetConvolution2dDescriptor(fc2_desc,
        0, 0,    // padding
        1, 1,    // stride
        1, 1,    // dilation
        HIPDNN_CROSS_CORRELATION,
        HIPDNN_DATA_FLOAT));
    
    // Enable Tensor Core operation for FC2
    CUDNN_CHECK(hipdnnSetConvolutionMathType(fc2_desc, HIPDNN_TENSOR_OP_MATH));
    
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(fc2_output_desc, HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_FLOAT, fc2_dims.n, fc2_dims.c, fc2_dims.h, fc2_dims.w));
    
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(fc2_bias_desc, HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_FLOAT, 1, fc2_dims.c, 1, 1));

    // ReLU activation setup
    CUDNN_CHECK(hipdnnSetActivationDescriptor(relu_activation,
        HIPDNN_ACTIVATION_RELU,
        HIPDNN_NOT_PROPAGATE_NAN,
        0.0));

    // Find best algorithms and workspace size for all convolutions
    workspace_size = 0;
    conv1_algo = findBestConvAlgorithm(input_desc, conv1_filter_desc, conv1_desc, 
                                      conv1_output_desc, &workspace_size);
    conv2_algo = findBestConvAlgorithm(pool1_output_desc, conv2_filter_desc, conv2_desc, 
                                      conv2_output_desc, &workspace_size);
    fc1_algo = findBestConvAlgorithm(pool2_flat_desc, fc1_filter_desc, fc1_desc, 
                                    fc1_output_desc, &workspace_size);
    fc2_algo = findBestConvAlgorithm(fc1_output_desc, fc2_filter_desc, fc2_desc, 
                                    fc2_output_desc, &workspace_size);

    // Allocate workspace memory
    CUDA_CHECK(hipMalloc(&d_workspace, workspace_size));

    // Allocate memory for layer outputs
    CUDA_CHECK(hipMalloc(&d_input, batch_size * 3 * 32 * 32 * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_conv1_output, batch_size * conv1_dims.c * conv1_dims.h * conv1_dims.w * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_pool1_output, batch_size * pool1_dims.c * pool1_dims.h * pool1_dims.w * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_conv2_output, batch_size * conv2_dims.c * conv2_dims.h * conv2_dims.w * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_pool2_output, batch_size * pool2_dims.c * pool2_dims.h * pool2_dims.w * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_fc1_output, batch_size * fc1_dims.c * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_fc2_output, batch_size * fc2_dims.c * sizeof(float)));

    std::cout << "Layer initialization complete." << std::endl;
    std::cout << "Workspace size: " << workspace_size / (1024.0 * 1024.0) << " MB" << std::endl;
}


void TensorCoreCNNInference::checkTensorCoreUsage() {
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, 0));
    
    std::cout << "\nGPU Configuration:" << std::endl;
    std::cout << "GPU: " << prop.name << std::endl;
    std::cout << "Compute Capability: " << prop.major << "." << prop.minor << std::endl;
    
    // Tensor Cores are available on:
    // - Volta (7.0) and above for FP16
    // - Ampere (8.0) and above for TF32
    bool hasTensorCores = false;
    bool supportsTF32 = false;
    
    if (prop.major >= 7) {
        hasTensorCores = true;
        if (prop.major >= 8) {
            supportsTF32 = true;
        }
    }
    
    std::cout << "Tensor Cores Available: " << (hasTensorCores ? "Yes" : "No") << std::endl;
    if (hasTensorCores) {
        std::cout << "TF32 Support: " << (supportsTF32 ? "Yes" : "No") << std::endl;
    }
    
    // Check math type configuration for each convolution
    hipdnnMathType_t mathType;
    CUDNN_CHECK(cudnnGetConvolutionMathType(conv1_desc, &mathType));
    std::cout << "\nConvolution Layer Math Types:" << std::endl;
    std::cout << "Conv1: " << 
        (mathType == HIPDNN_TENSOR_OP_MATH ? "Tensor Core" : "Standard") << std::endl;
    
    CUDNN_CHECK(cudnnGetConvolutionMathType(conv2_desc, &mathType));
    std::cout << "Conv2: " << 
        (mathType == HIPDNN_TENSOR_OP_MATH ? "Tensor Core" : "Standard") << std::endl;
    
    CUDNN_CHECK(cudnnGetConvolutionMathType(fc1_desc, &mathType));
    std::cout << "FC1: " << 
        (mathType == HIPDNN_TENSOR_OP_MATH ? "Tensor Core" : "Standard") << std::endl;
    
    CUDNN_CHECK(cudnnGetConvolutionMathType(fc2_desc, &mathType));
    std::cout << "FC2: " << 
        (mathType == HIPDNN_TENSOR_OP_MATH ? "Tensor Core" : "Standard") << std::endl;
    
    std::cout << "\nWorkspace Size: " << workspace_size / (1024.0 * 1024.0) << " MB" << std::endl;
}

void TensorCoreCNNInference::infer(const std::vector<float>& input_data) {
    const float alpha = 1.0f;
    const float beta = 0.0f;

    // Copy input to device
    CUDA_CHECK(hipMemcpy(d_input, input_data.data(), 
                         input_data.size() * sizeof(float), 
                         hipMemcpyHostToDevice));

    // Conv1 + ReLU
// Conv1 + ReLU
    CUDNN_CHECK(hipdnnConvolutionForward(cudnn, &alpha, 
        input_desc, d_input,
        conv1_filter_desc, d_conv1_weight,
        conv1_desc, conv1_algo,
        d_workspace, workspace_size,
        &beta, conv1_output_desc, d_conv1_output));
    
    // Add bias
    CUDNN_CHECK(hipdnnAddTensor(cudnn, 
        &alpha,
        conv1_bias_desc, d_conv1_bias,
        &alpha,
        conv1_output_desc, d_conv1_output));
    
    // ReLU activation
    CUDNN_CHECK(hipdnnActivationForward(cudnn, relu_activation,
        &alpha, conv1_output_desc, d_conv1_output,
        &beta, conv1_output_desc, d_conv1_output));

    // MaxPool1
    CUDNN_CHECK(hipdnnPoolingForward(cudnn, pooling_desc,
        &alpha, conv1_output_desc, d_conv1_output,
        &beta, pool1_output_desc, d_pool1_output));

    // Conv2 + ReLU
    CUDNN_CHECK(hipdnnConvolutionForward(cudnn, &alpha,
        pool1_output_desc, d_pool1_output,
        conv2_filter_desc, d_conv2_weight,
        conv2_desc, conv2_algo,
        d_workspace, workspace_size,
        &beta, conv2_output_desc, d_conv2_output));
    
    // Add bias
    CUDNN_CHECK(hipdnnAddTensor(cudnn, 
        &alpha,
        conv2_bias_desc, d_conv2_bias,
        &alpha,
        conv2_output_desc, d_conv2_output));
    
    // ReLU activation
    CUDNN_CHECK(hipdnnActivationForward(cudnn, relu_activation,
        &alpha, conv2_output_desc, d_conv2_output,
        &beta, conv2_output_desc, d_conv2_output));

    // MaxPool2
    CUDNN_CHECK(hipdnnPoolingForward(cudnn, pooling_desc,
        &alpha, conv2_output_desc, d_conv2_output,
        &beta, pool2_output_desc, d_pool2_output));

    // FC1 + ReLU
    CUDNN_CHECK(hipdnnConvolutionForward(cudnn, &alpha,
        pool2_flat_desc, d_pool2_output,
        fc1_filter_desc, d_fc1_weight,
        fc1_desc, fc1_algo,
        d_workspace, workspace_size,
        &beta, fc1_output_desc, d_fc1_output));
    
    // Add bias
    CUDNN_CHECK(hipdnnAddTensor(cudnn, 
        &alpha,
        fc1_bias_desc, d_fc1_bias,
        &alpha,
        fc1_output_desc, d_fc1_output));
    
    // ReLU activation
    CUDNN_CHECK(hipdnnActivationForward(cudnn, relu_activation,
        &alpha, fc1_output_desc, d_fc1_output,
        &beta, fc1_output_desc, d_fc1_output));

    // FC2 (final layer)
    CUDNN_CHECK(hipdnnConvolutionForward(cudnn, &alpha,
        fc1_output_desc, d_fc1_output,
        fc2_filter_desc, d_fc2_weight,
        fc2_desc, fc2_algo,
        d_workspace, workspace_size,
        &beta, fc2_output_desc, d_fc2_output));
    
    // Add final bias
    CUDNN_CHECK(hipdnnAddTensor(cudnn, 
        &alpha,
        fc2_bias_desc, d_fc2_bias,
        &alpha,
        fc2_output_desc, d_fc2_output));

    // Check for any CUDA errors
    CUDA_CHECK(hipGetLastError());
}

std::vector<float> TensorCoreCNNInference::getOutput() {
    std::vector<float> output(10);
    
    // Copy the output from device to host
    CUDA_CHECK(hipMemcpy(output.data(), d_fc2_output, 
                         output.size() * sizeof(float), 
                         hipMemcpyDeviceToHost));
    
    // Apply softmax normalization
    float max_val = *std::max_element(output.begin(), output.end());
    float sum = 0.0f;
    
    for (float& val : output) {
        val = std::exp(val - max_val);
        sum += val;
    }
    
    for (float& val : output) {
        val /= sum;
    }
    
    return output;
}

void TensorCoreCNNInference::evaluate(const std::vector<std::vector<float>>& images, 
                                    const std::vector<int>& labels) {
    // Create CUDA events for timing
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    
    size_t correct_count = 0;
    float total_time = 0.0f;
    
    std::cout << "\nStarting evaluation..." << std::endl;
    
    for (size_t i = 0; i < images.size(); ++i) {
        CUDA_CHECK(hipEventRecord(start));
        
        try {
            infer(images[i]);
            std::vector<float> output = getOutput();
            
            CUDA_CHECK(hipEventRecord(stop));
            CUDA_CHECK(hipEventSynchronize(stop));
            
            float milliseconds = 0;
            CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
            total_time += milliseconds;
            
            int predicted_label = std::distance(output.begin(), 
                                              std::max_element(output.begin(), output.end()));
            
            if (i % 100 == 0) {
                std::cout << "Processing image " << i << "..." << std::endl;
                std::cout << "Top 5 predictions:" << std::endl;
                
                std::vector<std::pair<int, float>> scores;
                for (size_t j = 0; j < output.size(); ++j) {
                    scores.emplace_back(j, output[j]);
                }
                
                std::sort(scores.begin(), scores.end(),
                         [](const auto& a, const auto& b) { return a.second > b.second; });
                
                for (int k = 0; k < std::min(5, static_cast<int>(scores.size())); ++k) {
                    std::cout << "  Class " << scores[k].first 
                             << ": " << std::fixed << std::setprecision(4) 
                             << scores[k].second * 100.0f << "%" << std::endl;
                }
                
                std::cout << "True label: " << labels[i] << std::endl;
                std::cout << "Inference time: " << milliseconds << " ms" << std::endl;
            }
            
            if (predicted_label == labels[i]) {
                ++correct_count;
            }
        }
        catch (const std::exception& e) {
            std::cerr << "Error processing image " << i << ": " << e.what() << std::endl;
            continue;
        }
    }

    float accuracy = static_cast<float>(correct_count) / images.size() * 100.0f;
    float avg_time = total_time / images.size();
    float throughput = 1000.0f / avg_time;  // images per second
    
    std::cout << "\nEvaluation Results:" << std::endl;
    std::cout << "Accuracy: " << std::fixed << std::setprecision(2) << accuracy << "%" << std::endl;
    std::cout << "Average inference time: " << std::fixed << std::setprecision(3) 
              << avg_time << " ms" << std::endl;
    std::cout << "Throughput: " << std::fixed << std::setprecision(1) 
              << throughput << " images/second" << std::endl;
    
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));
}

TensorCoreCNNInference::~TensorCoreCNNInference() {
    // Free device memory
    hipFree(d_input);
    hipFree(d_conv1_weight);
    hipFree(d_conv1_bias);
    hipFree(d_conv1_output);
    hipFree(d_pool1_output);
    hipFree(d_conv2_weight);
    hipFree(d_conv2_bias);
    hipFree(d_conv2_output);
    hipFree(d_pool2_output);
    hipFree(d_fc1_weight);
    hipFree(d_fc1_bias);
    hipFree(d_fc1_output);
    hipFree(d_fc2_weight);
    hipFree(d_fc2_bias);
    hipFree(d_fc2_output);
    hipFree(d_workspace);

    // Destroy descriptors
    hipdnnDestroyTensorDescriptor(input_desc);
    hipdnnDestroyTensorDescriptor(conv1_output_desc);
    hipdnnDestroyTensorDescriptor(pool1_output_desc);
    hipdnnDestroyTensorDescriptor(conv2_output_desc);
    hipdnnDestroyTensorDescriptor(pool2_output_desc);
    hipdnnDestroyTensorDescriptor(pool2_flat_desc);
    hipdnnDestroyTensorDescriptor(fc1_output_desc);
    hipdnnDestroyTensorDescriptor(fc2_output_desc);
    
    hipdnnDestroyFilterDescriptor(conv1_filter_desc);
    hipdnnDestroyFilterDescriptor(conv2_filter_desc);
    hipdnnDestroyFilterDescriptor(fc1_filter_desc);
    hipdnnDestroyFilterDescriptor(fc2_filter_desc);
    
    hipdnnDestroyTensorDescriptor(conv1_bias_desc);
    hipdnnDestroyTensorDescriptor(conv2_bias_desc);
    hipdnnDestroyTensorDescriptor(fc1_bias_desc);
    hipdnnDestroyTensorDescriptor(fc2_bias_desc);
    
    hipdnnDestroyConvolutionDescriptor(conv1_desc);
    hipdnnDestroyConvolutionDescriptor(conv2_desc);
    hipdnnDestroyConvolutionDescriptor(fc1_desc);
    hipdnnDestroyConvolutionDescriptor(fc2_desc);
    
    hipdnnDestroyActivationDescriptor(relu_activation);
    hipdnnDestroyPoolingDescriptor(pooling_desc);
    
    hipdnnDestroy(cudnn);
}

void TensorCoreCNNInference::loadWeights() {
    std::cout << "Loading model weights..." << std::endl;
    
    // Load weights from binary files
    auto conv1_weights = loadBinaryFile<float>("../../../data/weights/conv1.weight_fp32.bin");
    auto conv1_biases = loadBinaryFile<float>("../../../data/weights/conv1.bias_fp32.bin");
    auto conv2_weights = loadBinaryFile<float>("../../../data/weights/conv2.weight_fp32.bin");
    auto conv2_biases = loadBinaryFile<float>("../../../data/weights/conv2.bias_fp32.bin");
    auto fc1_weights = loadBinaryFile<float>("../../../data/weights/fc1.weight_fp32.bin");
    auto fc1_biases = loadBinaryFile<float>("../../../data/weights/fc1.bias_fp32.bin");
    auto fc2_weights = loadBinaryFile<float>("../../../data/weights/fc2.weight_fp32.bin");
    auto fc2_biases = loadBinaryFile<float>("../../../data/weights/fc2.bias_fp32.bin");
    
    // Verify sizes
    const size_t conv1_weights_size = 32 * 3 * 3 * 3;
    const size_t conv1_bias_size = 32;
    const size_t conv2_weights_size = 64 * 32 * 3 * 3;
    const size_t conv2_bias_size = 64;
    const size_t fc1_weights_size = 128 * (64 * 8 * 8);
    const size_t fc1_bias_size = 128;
    const size_t fc2_weights_size = 10 * 128;
    const size_t fc2_bias_size = 10;
    
    // Verify sizes match expected dimensions
    if (conv1_weights.size() != conv1_weights_size ||
        conv1_biases.size() != conv1_bias_size ||
        conv2_weights.size() != conv2_weights_size ||
        conv2_biases.size() != conv2_bias_size ||
        fc1_weights.size() != fc1_weights_size ||
        fc1_biases.size() != fc1_bias_size ||
        fc2_weights.size() != fc2_weights_size ||
        fc2_biases.size() != fc2_bias_size) {
        throw std::runtime_error("Weight file sizes do not match expected dimensions");
    }
    
    // Allocate and copy weights to device
    CUDA_CHECK(hipMalloc(&d_conv1_weight, conv1_weights_size * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_conv1_bias, conv1_bias_size * sizeof(float)));
    CUDA_CHECK(hipMemcpy(d_conv1_weight, conv1_weights.data(), 
                         conv1_weights_size * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_conv1_bias, conv1_biases.data(), 
                         conv1_bias_size * sizeof(float), hipMemcpyHostToDevice));
    
CUDA_CHECK(hipMalloc(&d_conv2_weight, conv2_weights_size * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_conv2_bias, conv2_bias_size * sizeof(float)));
    CUDA_CHECK(hipMemcpy(d_conv2_weight, conv2_weights.data(),
                         conv2_weights_size * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_conv2_bias, conv2_biases.data(),
                         conv2_bias_size * sizeof(float), hipMemcpyHostToDevice));

    CUDA_CHECK(hipMalloc(&d_fc1_weight, fc1_weights_size * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_fc1_bias, fc1_bias_size * sizeof(float)));
    CUDA_CHECK(hipMemcpy(d_fc1_weight, fc1_weights.data(),
                         fc1_weights_size * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_fc1_bias, fc1_biases.data(),
                         fc1_bias_size * sizeof(float), hipMemcpyHostToDevice));

    CUDA_CHECK(hipMalloc(&d_fc2_weight, fc2_weights_size * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_fc2_bias, fc2_bias_size * sizeof(float)));
    CUDA_CHECK(hipMemcpy(d_fc2_weight, fc2_weights.data(),
                         fc2_weights_size * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_fc2_bias, fc2_biases.data(),
                         fc2_bias_size * sizeof(float), hipMemcpyHostToDevice));

    // Verify weights were loaded successfully
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        throw std::runtime_error(std::string("CUDA error while loading weights: ") +
                               hipGetErrorString(error));
    }

    std::cout << "Successfully loaded all weights to GPU." << std::endl;
    
    // Print first few weights for verification
    std::cout << "Conv1 weights first values: ";
    for (int i = 0; i < 5; i++) {
        std::cout << std::fixed << std::setprecision(6) << conv1_weights[i] << " ";
    }
    std::cout << std::endl;
}

int main() {
    try {
        std::cout << "Loading validation data..." << std::endl;
        auto validation_images = loadBinaryFile<float>("../../../data/validation/validation_images.bin");
        auto validation_labels = loadBinaryFile<int>("../../../data/validation/validation_labels.bin");

        std::vector<std::vector<float>> images;
        const size_t image_size = 3 * 32 * 32;
        for (size_t i = 0; i < validation_images.size(); i += image_size) {
            images.push_back(std::vector<float>(
                validation_images.begin() + i,
                validation_images.begin() + i + image_size));
        }

        std::cout << "Creating TensorCore inference engine..." << std::endl;
        TensorCoreCNNInference cnn_inference;
        
        std::cout << "Running evaluation on validation data..." << std::endl;
        cnn_inference.evaluate(images, validation_labels);

    } catch (const std::exception& e) {
        std::cerr << "Error: " << e.what() << std::endl;
        return 1;
    }

    return 0;
}