#include "hip/hip_runtime.h"
#include </usr/include/hipDNN.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <cassert>
#include <algorithm>
#include <chrono>
#include <iomanip>
#include <hip/hip_fp16.h>
#include <sstream>

#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)
template<typename T>
void check(T err, const char* const func, const char* const file,
           const int line) {
    if (err != hipSuccess) {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                  << std::endl;
        std::cerr << hipGetErrorString(err) << " " << func << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

// Load binary data from file
template <typename T>
std::vector<T> loadBinaryFile(const std::string& filename) {
    std::ifstream file(filename, std::ios::binary);
    if (!file.is_open()) {
        std::cerr << "Error: Could not open file " << filename << std::endl;
        std::exit(EXIT_FAILURE);
    }
    
    // Get file size in bytes
    file.seekg(0, std::ios::end);
    size_t file_size = file.tellg();
    file.seekg(0, std::ios::beg);
    
    // Calculate number of elements
    size_t num_elements = file_size / sizeof(T);
    
    std::cout << "Loading " << filename << " - File size: " << file_size 
              << " bytes, Elements: " << num_elements << std::endl;
    
    std::vector<T> buffer(num_elements);
    file.read(reinterpret_cast<char*>(buffer.data()), file_size);
    file.close();
    
    return buffer;
}

// Helper functions for CUDA kernel error checking
#define CUDNN_CHECK(call)                                                         \
    {                                                                             \
        hipdnnStatus_t err = call;                                                 \
        if (err != HIPDNN_STATUS_SUCCESS) {                                        \
            std::cerr << "CuDNN Error: " << hipdnnGetErrorString(err) << std::endl; \
            std::exit(EXIT_FAILURE);                                              \
        }                                                                         \
    }

#define CUDA_CHECK(call)                                                         \
    {                                                                            \
        hipError_t err = call;                                                  \
        if (err != hipSuccess) {                                                \
            std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl; \
            std::exit(EXIT_FAILURE);                                             \
        }                                                                        \
    }

// CUDA kernels for type conversion
__global__ void floatToHalf(float* input, half* output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = __float2half(input[idx]);
    }
}

__global__ void halfToFloat(half* input, float* output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = __half2float(input[idx]);
    }
}

class CUDACNNInference {
public:
    CUDACNNInference();
    ~CUDACNNInference();
    void loadWeights();
    void initializeLayers();
    void infer(const std::vector<float>& input_data);
    std::vector<float> getOutput();

private:
    hipdnnHandle_t cudnn;
    
    // Layer descriptors
    hipdnnTensorDescriptor_t input_desc;
    hipdnnTensorDescriptor_t conv1_output_desc;
    hipdnnTensorDescriptor_t pool1_output_desc;
    hipdnnTensorDescriptor_t conv2_output_desc;
    hipdnnTensorDescriptor_t pool2_output_desc;
    hipdnnTensorDescriptor_t pool2_flat_desc;
    hipdnnTensorDescriptor_t fc1_input_desc;
    hipdnnTensorDescriptor_t fc1_output_desc;
    hipdnnTensorDescriptor_t fc2_input_desc;
    hipdnnTensorDescriptor_t fc2_output_desc;
    hipdnnFilterDescriptor_t fc1_weight_desc;
    hipdnnFilterDescriptor_t fc2_weight_desc;
    
    // Bias descriptors
    hipdnnTensorDescriptor_t conv1_bias_desc;
    hipdnnTensorDescriptor_t conv2_bias_desc;
    hipdnnTensorDescriptor_t fc1_bias_desc;
    hipdnnTensorDescriptor_t fc2_bias_desc;
    
    // Filter and convolution descriptors
    hipdnnFilterDescriptor_t conv1_filter_desc;
    hipdnnFilterDescriptor_t conv2_filter_desc;
    hipdnnConvolutionDescriptor_t conv1_desc;
    hipdnnConvolutionDescriptor_t conv2_desc;
    hipdnnConvolutionDescriptor_t fc1_desc;
    hipdnnConvolutionDescriptor_t fc2_desc;
    
    // Activation and pooling descriptors
    hipdnnActivationDescriptor_t relu_activation;
    hipdnnPoolingDescriptor_t pooling_desc;

    int fc1_input_size;

    // Device memory pointers
    float *d_input;
    half *d_conv1_weight, *d_conv1_bias, *d_conv1_output;
    half *d_pool1_output;
    half *d_conv2_weight, *d_conv2_bias, *d_conv2_output;
    half *d_pool2_output;
    half *d_fc1_weight, *d_fc1_bias, *d_fc1_output;
    half *d_fc2_weight, *d_fc2_bias, *d_fc2_output;

    size_t workspace_size;
    void *d_workspace;

    int batch_size = 1;
    struct LayerDims {
        int n, c, h, w;
    };
    LayerDims conv1_dims, pool1_dims, conv2_dims, pool2_dims, fc1_dims, fc2_dims;
};

CUDACNNInference::CUDACNNInference() {
    std::cout << "Initializing CuDNN..." << std::endl;
    CUDNN_CHECK(hipdnnCreate(&cudnn));

    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&input_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&conv1_output_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&pool1_output_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&conv2_output_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&pool2_output_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&pool2_flat_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&fc1_input_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&fc1_output_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&fc2_input_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&fc2_output_desc));
    CUDNN_CHECK(hipdnnCreateFilterDescriptor(&fc1_weight_desc));
    CUDNN_CHECK(hipdnnCreateFilterDescriptor(&fc2_weight_desc));

    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&conv1_bias_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&conv2_bias_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&fc1_bias_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&fc2_bias_desc));
    
    CUDNN_CHECK(hipdnnCreateFilterDescriptor(&conv1_filter_desc));
    CUDNN_CHECK(hipdnnCreateFilterDescriptor(&conv2_filter_desc));
    CUDNN_CHECK(hipdnnCreateConvolutionDescriptor(&conv1_desc));
    CUDNN_CHECK(hipdnnCreateConvolutionDescriptor(&conv2_desc));
    CUDNN_CHECK(hipdnnCreateConvolutionDescriptor(&fc1_desc));
    CUDNN_CHECK(hipdnnCreateConvolutionDescriptor(&fc2_desc));
    
    CUDNN_CHECK(hipdnnCreateActivationDescriptor(&relu_activation));
    CUDNN_CHECK(hipdnnCreatePoolingDescriptor(&pooling_desc));

    loadWeights();
    initializeLayers();
}

void CUDACNNInference::loadWeights() {
    std::cout << "Loading FP16 model weights..." << std::endl;
    
    // Load weights directly as FP16
    auto conv1_weights = loadBinaryFile<half>("../../data/weights/conv1.weight_fp16.bin");
    auto conv1_biases = loadBinaryFile<half>("../../data/weights/conv1.bias_fp16.bin");
    auto conv2_weights = loadBinaryFile<half>("../../data/weights/conv2.weight_fp16.bin");
    auto conv2_biases = loadBinaryFile<half>("../../data/weights/conv2.bias_fp16.bin");
    auto fc1_weights = loadBinaryFile<half>("../../data/weights/fc1.weight_fp16.bin");
    auto fc1_biases = loadBinaryFile<half>("../../data/weights/fc1.bias_fp16.bin");
    auto fc2_weights = loadBinaryFile<half>("../../data/weights/fc2.weight_fp16.bin");
    auto fc2_biases = loadBinaryFile<half>("../../data/weights/fc2.bias_fp16.bin");

    // Verify sizes based on model architecture
    const size_t conv1_weights_size = 32 * 3 * 3 * 3;
    const size_t conv1_bias_size = 32;
    const size_t conv2_weights_size = 64 * 32 * 3 * 3;
    const size_t conv2_bias_size = 64;
    const size_t fc1_weights_size = 128 * (64 * 8 * 8);
    const size_t fc1_bias_size = 128;
    const size_t fc2_weights_size = 10 * 128;
    const size_t fc2_bias_size = 10;
    
    // Allocate and copy weights to device
    CUDA_CHECK(hipMalloc(&d_conv1_weight, conv1_weights_size * sizeof(half)));
    CUDA_CHECK(hipMalloc(&d_conv1_bias, conv1_bias_size * sizeof(half)));
    CUDA_CHECK(hipMemcpy(d_conv1_weight, conv1_weights.data(), 
                         conv1_weights_size * sizeof(half), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_conv1_bias, conv1_biases.data(), 
                         conv1_bias_size * sizeof(half), hipMemcpyHostToDevice));
    
    CUDA_CHECK(hipMalloc(&d_conv2_weight, conv2_weights_size * sizeof(half)));
    CUDA_CHECK(hipMalloc(&d_conv2_bias, conv2_bias_size * sizeof(half)));
    CUDA_CHECK(hipMemcpy(d_conv2_weight, conv2_weights.data(),
                         conv2_weights_size * sizeof(half), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_conv2_bias, conv2_biases.data(),
                         conv2_bias_size * sizeof(half), hipMemcpyHostToDevice));

    CUDA_CHECK(hipMalloc(&d_fc1_weight, fc1_weights_size * sizeof(half)));
    CUDA_CHECK(hipMalloc(&d_fc1_bias, fc1_bias_size * sizeof(half)));
    CUDA_CHECK(hipMemcpy(d_fc1_weight, fc1_weights.data(),
                         fc1_weights_size * sizeof(half), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_fc1_bias, fc1_biases.data(),
                         fc1_bias_size * sizeof(half), hipMemcpyHostToDevice));

    CUDA_CHECK(hipMalloc(&d_fc2_weight, fc2_weights_size * sizeof(half)));
    CUDA_CHECK(hipMalloc(&d_fc2_bias, fc2_bias_size * sizeof(half)));
    CUDA_CHECK(hipMemcpy(d_fc2_weight, fc2_weights.data(),
                         fc2_weights_size * sizeof(half), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_fc2_bias, fc2_biases.data(),
                         fc2_bias_size * sizeof(half), hipMemcpyHostToDevice));

    std::cout << "Successfully loaded all weights to GPU." << std::endl;
}

void CUDACNNInference::initializeLayers() {
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(input_desc, HIPDNN_TENSOR_NCHW, 
        HIPDNN_DATA_HALF, batch_size, 3, 32, 32));

    CUDNN_CHECK(hipdnnSetFilter4dDescriptor(conv1_filter_desc, HIPDNN_DATA_HALF, 
        HIPDNN_TENSOR_NCHW, 32, 3, 3, 3));
    
    CUDNN_CHECK(hipdnnSetConvolution2dDescriptor(conv1_desc, 
        1, 1,    // padding
        1, 1,    // stride
        1, 1,    // dilation
        HIPDNN_CROSS_CORRELATION, 
        HIPDNN_DATA_HALF));
    CUDNN_CHECK(hipdnnSetConvolutionMathType(conv1_desc, HIPDNN_DEFAULT_MATH));

    CUDNN_CHECK(hipdnnGetConvolution2dForwardOutputDim(conv1_desc, input_desc, 
        conv1_filter_desc, &conv1_dims.n, &conv1_dims.c, &conv1_dims.h, &conv1_dims.w));
    
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(conv1_output_desc, HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_HALF, conv1_dims.n, conv1_dims.c, conv1_dims.h, conv1_dims.w));
    
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(conv1_bias_desc, HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_HALF, 1, conv1_dims.c, 1, 1));

    CUDNN_CHECK(hipdnnSetPooling2dDescriptor(pooling_desc, 
        HIPDNN_POOLING_MAX,
        HIPDNN_NOT_PROPAGATE_NAN,
        2, 2,    // window size
        0, 0,    // padding
        2, 2));  // stride

    CUDNN_CHECK(hipdnnGetPooling2dForwardOutputDim(pooling_desc,
        conv1_output_desc,
        &pool1_dims.n, &pool1_dims.c, &pool1_dims.h, &pool1_dims.w));
    
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(pool1_output_desc, HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_HALF, pool1_dims.n, pool1_dims.c, pool1_dims.h, pool1_dims.w));

    CUDNN_CHECK(hipdnnSetFilter4dDescriptor(conv2_filter_desc, HIPDNN_DATA_HALF, 
        HIPDNN_TENSOR_NCHW, 64, 32, 3, 3));
    
    CUDNN_CHECK(hipdnnSetConvolution2dDescriptor(conv2_desc, 
        1, 1,    // padding
        1, 1,    // stride
        1, 1,    // dilation
        HIPDNN_CROSS_CORRELATION, 
        HIPDNN_DATA_HALF));
    CUDNN_CHECK(hipdnnSetConvolutionMathType(conv2_desc, HIPDNN_DEFAULT_MATH));
    
    CUDNN_CHECK(hipdnnGetConvolution2dForwardOutputDim(conv2_desc, pool1_output_desc, 
        conv2_filter_desc, &conv2_dims.n, &conv2_dims.c, &conv2_dims.h, &conv2_dims.w));
    
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(conv2_output_desc, HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_HALF, conv2_dims.n, conv2_dims.c, conv2_dims.h, conv2_dims.w));
    
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(conv2_bias_desc, HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_HALF, 1, conv2_dims.c, 1, 1));

    pool2_dims = {conv2_dims.n, conv2_dims.c, conv2_dims.h/2, conv2_dims.w/2};
    
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(pool2_output_desc, HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_HALF, pool2_dims.n, pool2_dims.c, pool2_dims.h, pool2_dims.w));

    fc1_input_size = pool2_dims.c * pool2_dims.h * pool2_dims.w;

    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(pool2_flat_desc, HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_HALF, batch_size, fc1_input_size, 1, 1));

    CUDNN_CHECK(hipdnnSetFilter4dDescriptor(fc1_weight_desc, HIPDNN_DATA_HALF,
        HIPDNN_TENSOR_NCHW, 128, fc1_input_size, 1, 1));
    
    CUDNN_CHECK(hipdnnSetConvolution2dDescriptor(fc1_desc,
        0, 0,    // padding
        1, 1,    // stride
        1, 1,    // dilation
        HIPDNN_CROSS_CORRELATION,
        HIPDNN_DATA_HALF));
    CUDNN_CHECK(hipdnnSetConvolutionMathType(fc1_desc, HIPDNN_DEFAULT_MATH));

    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(fc1_output_desc, HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_HALF, batch_size, 128, 1, 1));
    
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(fc1_bias_desc, HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_HALF, 1, 128, 1, 1));

    CUDNN_CHECK(hipdnnSetFilter4dDescriptor(fc2_weight_desc, HIPDNN_DATA_HALF,
        HIPDNN_TENSOR_NCHW, 10, 128, 1, 1));
    
    CUDNN_CHECK(hipdnnSetConvolution2dDescriptor(fc2_desc,
        0, 0,    // padding
        1, 1,    // stride
        1, 1,    // dilation
        HIPDNN_CROSS_CORRELATION,
        HIPDNN_DATA_HALF));
    CUDNN_CHECK(hipdnnSetConvolutionMathType(fc2_desc, HIPDNN_DEFAULT_MATH));

    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(fc2_output_desc, HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_HALF, batch_size, 10, 1, 1));
    
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(fc2_bias_desc, HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_HALF, 1, 10, 1, 1));

    CUDNN_CHECK(hipdnnSetActivationDescriptor(relu_activation,
        HIPDNN_ACTIVATION_RELU,
        HIPDNN_NOT_PROPAGATE_NAN,
        0.0));

    size_t workspace_sizes[4];
    CUDNN_CHECK(hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
        input_desc, conv1_filter_desc, conv1_desc, conv1_output_desc,
        HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM, &workspace_sizes[0]));
    
    CUDNN_CHECK(hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
        pool1_output_desc, conv2_filter_desc, conv2_desc, conv2_output_desc,
        HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM, &workspace_sizes[1]));
    
    CUDNN_CHECK(hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
        pool2_flat_desc, fc1_weight_desc, fc1_desc, fc1_output_desc,
        HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM, &workspace_sizes[2]));
    
    CUDNN_CHECK(hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
        fc1_output_desc, fc2_weight_desc, fc2_desc, fc2_output_desc,
        HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM, &workspace_sizes[3]));

    workspace_size = *std::max_element(workspace_sizes, workspace_sizes + 4);
    CUDA_CHECK(hipMalloc(&d_workspace, workspace_size));

    size_t input_bytes = batch_size * 3 * 32 * 32 * sizeof(float);
    size_t conv1_output_bytes = batch_size * conv1_dims.c * conv1_dims.h * conv1_dims.w * sizeof(half);
    size_t pool1_output_bytes = batch_size * pool1_dims.c * pool1_dims.h * pool1_dims.w * sizeof(half);
    size_t conv2_output_bytes = batch_size * conv2_dims.c * conv2_dims.h * conv2_dims.w * sizeof(half);
    size_t pool2_output_bytes = batch_size * pool2_dims.c * pool2_dims.h * pool2_dims.w * sizeof(half);
    size_t fc1_output_bytes = batch_size * 128 * sizeof(half);
    size_t fc2_output_bytes = batch_size * 10 * sizeof(half);

    CUDA_CHECK(hipMalloc(&d_input, input_bytes));
    CUDA_CHECK(hipMalloc(&d_conv1_output, conv1_output_bytes));
    CUDA_CHECK(hipMalloc(&d_pool1_output, pool1_output_bytes));
    CUDA_CHECK(hipMalloc(&d_conv2_output, conv2_output_bytes));
    CUDA_CHECK(hipMalloc(&d_pool2_output, pool2_output_bytes));
    CUDA_CHECK(hipMalloc(&d_fc1_output, fc1_output_bytes));
    CUDA_CHECK(hipMalloc(&d_fc2_output, fc2_output_bytes));

    std::cout << "Layer initialization complete." << std::endl;
    std::cout << "Workspace size: " << workspace_size << " bytes" << std::endl;
}

void CUDACNNInference::infer(const std::vector<float>& input_data) {
    const float alpha_f = 1.0f;
    const float beta_f = 0.0f;
    const void* alpha_ptr = &alpha_f;
    const void* beta_ptr = &beta_f;
    
    size_t expected_input_size = batch_size * 3 * 32 * 32;
    if (input_data.size() != expected_input_size) {
        throw std::runtime_error("Input data size mismatch");
    }

    float* d_input_float;
    CUDA_CHECK(hipMalloc(&d_input_float, input_data.size() * sizeof(float)));
    CUDA_CHECK(hipMemcpy(d_input_float, input_data.data(), input_data.size() * sizeof(float), hipMemcpyHostToDevice));

    int blockSize = 256;
    int numBlocks = (input_data.size() + blockSize - 1) / blockSize;
    floatToHalf<<<numBlocks, blockSize>>>(d_input_float, (half*)d_input, input_data.size());
    CHECK_CUDA_ERROR(hipGetLastError());
    CUDA_CHECK(hipFree(d_input_float));

    CUDNN_CHECK(hipdnnConvolutionForward(cudnn, alpha_ptr, 
        input_desc, d_input,
        conv1_filter_desc, d_conv1_weight,
        conv1_desc, HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM,
        d_workspace, workspace_size,
        beta_ptr, conv1_output_desc, d_conv1_output));
    
    CUDNN_CHECK(hipdnnAddTensor(cudnn, 
        alpha_ptr,
        conv1_bias_desc, d_conv1_bias,
        alpha_ptr,
        conv1_output_desc, d_conv1_output));
    
    CUDNN_CHECK(hipdnnActivationForward(cudnn, relu_activation,
        alpha_ptr, conv1_output_desc, d_conv1_output,
        beta_ptr, conv1_output_desc, d_conv1_output));

    CUDNN_CHECK(hipdnnPoolingForward(cudnn, pooling_desc,
        alpha_ptr, conv1_output_desc, d_conv1_output,
        beta_ptr, pool1_output_desc, d_pool1_output));

    CUDNN_CHECK(hipdnnConvolutionForward(cudnn, alpha_ptr,
        pool1_output_desc, d_pool1_output,
        conv2_filter_desc, d_conv2_weight,
        conv2_desc, HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM,
        d_workspace, workspace_size,
        beta_ptr, conv2_output_desc, d_conv2_output));
    
    CUDNN_CHECK(hipdnnAddTensor(cudnn, 
        alpha_ptr,
        conv2_bias_desc, d_conv2_bias,
        alpha_ptr,
        conv2_output_desc, d_conv2_output));
    
    CUDNN_CHECK(hipdnnActivationForward(cudnn, relu_activation,
        alpha_ptr, conv2_output_desc, d_conv2_output,
        beta_ptr, conv2_output_desc, d_conv2_output));

    CUDNN_CHECK(hipdnnPoolingForward(cudnn, pooling_desc,
        alpha_ptr, conv2_output_desc, d_conv2_output,
        beta_ptr, pool2_output_desc, d_pool2_output));

    CUDNN_CHECK(hipdnnConvolutionForward(cudnn, alpha_ptr,
        pool2_flat_desc, d_pool2_output,
        fc1_weight_desc, d_fc1_weight,
        fc1_desc, HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
        d_workspace, workspace_size,
        beta_ptr, fc1_output_desc, d_fc1_output));
    
    CUDNN_CHECK(hipdnnAddTensor(cudnn, 
        alpha_ptr,
        fc1_bias_desc, d_fc1_bias,
        alpha_ptr,
        fc1_output_desc, d_fc1_output));
    
    CUDNN_CHECK(hipdnnActivationForward(cudnn, relu_activation,
        alpha_ptr, fc1_output_desc, d_fc1_output,
        beta_ptr, fc1_output_desc, d_fc1_output));

    CUDNN_CHECK(hipdnnConvolutionForward(cudnn, alpha_ptr,
        fc1_output_desc, d_fc1_output,
        fc2_weight_desc, d_fc2_weight,
        fc2_desc, HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
        d_workspace, workspace_size,
        beta_ptr, fc2_output_desc, d_fc2_output));
    
    CUDNN_CHECK(hipdnnAddTensor(cudnn, 
        alpha_ptr,
        fc2_bias_desc, d_fc2_bias,
        alpha_ptr,
        fc2_output_desc, d_fc2_output));

    CHECK_CUDA_ERROR(hipGetLastError());
}

std::vector<float> CUDACNNInference::getOutput() {
    std::vector<float> output(10);
    
    float* d_output_float;
    CUDA_CHECK(hipMalloc(&d_output_float, 10 * sizeof(float)));
    
    int blockSize = 256;
    int numBlocks = (10 + blockSize - 1) / blockSize;
    halfToFloat<<<numBlocks, blockSize>>>((half*)d_fc2_output, d_output_float, 10);
    CHECK_CUDA_ERROR(hipGetLastError());
    
    CUDA_CHECK(hipMemcpy(output.data(), d_output_float, output.size() * sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipFree(d_output_float));
    
    // Apply softmax normalization
    float max_val = *std::max_element(output.begin(), output.end());
    float sum = 0.0f;
    
    for (float& val : output) {
        val = std::exp(val - max_val);
        sum += val;
    }
    
    for (float& val : output) {
        val /= sum;
    }
    
    return output;
}

CUDACNNInference::~CUDACNNInference() {
    hipFree(d_input);
    hipFree(d_conv1_weight);
    hipFree(d_conv1_bias);
    hipFree(d_conv1_output);
    hipFree(d_pool1_output);
    hipFree(d_conv2_weight);
    hipFree(d_conv2_bias);
    hipFree(d_conv2_output);
    hipFree(d_pool2_output);
    hipFree(d_fc1_weight);
    hipFree(d_fc1_bias);
    hipFree(d_fc1_output);
    hipFree(d_fc2_weight);
    hipFree(d_fc2_bias);
    hipFree(d_fc2_output);
    hipFree(d_workspace);

    hipdnnDestroyTensorDescriptor(input_desc);
    hipdnnDestroyTensorDescriptor(conv1_output_desc);
    hipdnnDestroyTensorDescriptor(pool1_output_desc);
    hipdnnDestroyTensorDescriptor(conv2_output_desc);
    hipdnnDestroyTensorDescriptor(pool2_output_desc);
    hipdnnDestroyTensorDescriptor(pool2_flat_desc);
    hipdnnDestroyTensorDescriptor(fc1_input_desc);
    hipdnnDestroyTensorDescriptor(fc1_output_desc);
    hipdnnDestroyTensorDescriptor(fc2_input_desc);
    hipdnnDestroyTensorDescriptor(fc2_output_desc);
    hipdnnDestroyFilterDescriptor(fc1_weight_desc);
    hipdnnDestroyFilterDescriptor(fc2_weight_desc);

    hipdnnDestroyTensorDescriptor(conv1_bias_desc);
    hipdnnDestroyTensorDescriptor(conv2_bias_desc);
    hipdnnDestroyTensorDescriptor(fc1_bias_desc);
    hipdnnDestroyTensorDescriptor(fc2_bias_desc);

    hipdnnDestroyFilterDescriptor(conv1_filter_desc);
    hipdnnDestroyFilterDescriptor(conv2_filter_desc);
    hipdnnDestroyConvolutionDescriptor(conv1_desc);
    hipdnnDestroyConvolutionDescriptor(conv2_desc);
    hipdnnDestroyConvolutionDescriptor(fc1_desc);
    hipdnnDestroyConvolutionDescriptor(fc2_desc);

    hipdnnDestroyActivationDescriptor(relu_activation);
    hipdnnDestroyPoolingDescriptor(pooling_desc);

    hipdnnDestroy(cudnn);
}

void parseArguments(int argc, char** argv, int& gpu_id, int& repeat_factor) {
    if (argc >= 3) {
        gpu_id = std::atoi(argv[1]);
        repeat_factor = std::atoi(argv[2]);
    } else {
        std::cerr << "Usage: " << argv[0] << " <gpu_id> <repeat_factor>" << std::endl;
        std::cerr << "Example: " << argv[0] << " 0 10" << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

int main(int argc, char** argv) {
    int gpu_id = 0;
    int repeat_factor = 1;

    // Parse GPU ID and repeat factor from arguments
    parseArguments(argc, argv, gpu_id, repeat_factor);

    // Set the GPU device at runtime
    CUDA_CHECK(hipSetDevice(gpu_id));

    std::cout << "Running on GPU: " << gpu_id << std::endl;
    std::cout << "Repeat factor: " << repeat_factor << std::endl;

    try {
        std::cout << "Loading validation data..." << std::endl;
        auto validation_images = loadBinaryFile<float>("../../data/validation/validation_images.bin");
        auto validation_labels = loadBinaryFile<int>("../../data/validation/validation_labels.bin");

        // Original image size for CIFAR-10 (3 channels, 32x32 resolution)
        size_t image_size = 3 * 32 * 32;

        // Organize the original data into individual images
        std::vector<std::vector<float>> images;
        for (size_t i = 0; i < validation_images.size(); i += image_size) {
            images.push_back(std::vector<float>(validation_images.begin() + i, 
                                             validation_images.begin() + i + image_size));
        }

        // Repeat the dataset
        std::vector<std::vector<float>> repeated_images;
        std::vector<int> repeated_labels;

        for (int i = 0; i < repeat_factor; ++i) {
            repeated_images.insert(repeated_images.end(), images.begin(), images.end());
            repeated_labels.insert(repeated_labels.end(), validation_labels.begin(), validation_labels.end());
        }
        
        int total_images = repeated_images.size();

        std::cout << "Total images after repeating: " << total_images << std::endl;

        std::cout << "Creating CUDA inference engine..." << std::endl;
        CUDACNNInference cnn;

        std::cout << "\n=== Starting Evaluation ===" << std::endl;
        std::cout << "Model type: CUDA Core FP16" << std::endl;

        // Create CUDA events for timing
        hipEvent_t start, stop;
        CUDA_CHECK(hipEventCreate(&start));
        CUDA_CHECK(hipEventCreate(&stop));

        size_t correct_count = 0;
        float total_time = 0.0f;

        // Warmup run
        std::cout << "Performing warmup runs..." << std::endl;
        for (int i = 0; i < 10; i++) {
            cnn.infer(repeated_images[0]);
        }

        // Main evaluation loop
        std::cout << "Starting main evaluation..." << std::endl;
        for (size_t i = 0; i < total_images; ++i) {
            try {
                CUDA_CHECK(hipEventRecord(start));
                
                cnn.infer(repeated_images[i]);
                std::vector<float> output = cnn.getOutput();
                
                CUDA_CHECK(hipEventRecord(stop));
                CUDA_CHECK(hipEventSynchronize(stop));
                
                float milliseconds = 0;
                CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
                total_time += milliseconds;

                int predicted_label = std::distance(output.begin(), 
                                                 std::max_element(output.begin(), output.end()));
                
                if (predicted_label == repeated_labels[i]) {
                    ++correct_count;
                }

                if (i % 1000 == 0) {
                    float running_accuracy = (static_cast<float>(correct_count) / (i + 1)) * 100.0f;
                    std::cout << "Processed " << i + 1 << "/" << total_images 
                            << " images. Running accuracy: " << std::fixed 
                            << std::setprecision(2) << running_accuracy << "%" << std::endl;
                }
            }
            catch (const std::exception& e) {
                std::cerr << "Error processing image " << i << ": " << e.what() << std::endl;
                continue;
            }
        }

        // Print final statistics
        float accuracy = static_cast<float>(correct_count) / total_images * 100.0f;
        float avg_time = total_time / total_images;
        float throughput = 1000.0f / avg_time;

        std::cout << "\n=== Final Results ===" << std::endl;
        std::cout << "Model type: CUDA Core FP16" << std::endl;
        std::cout << "Total images: " << total_images << std::endl;
        std::cout << "Correct predictions: " << correct_count << std::endl;
        std::cout << "Accuracy: " << std::fixed << std::setprecision(2) << accuracy << "%" << std::endl;
        std::cout << "Average inference time: " << std::fixed << std::setprecision(3) 
                << avg_time << " ms" << std::endl;
        std::cout << "Throughput: " << std::fixed << std::setprecision(1) 
                << throughput << " images/second" << std::endl;
        std::cout << "Total evaluation time: " << std::fixed << std::setprecision(2) 
                << total_time / 1000.0f << " seconds" << std::endl;

        CUDA_CHECK(hipEventDestroy(start));
        CUDA_CHECK(hipEventDestroy(stop));

    } catch (const std::exception& e) {
        std::cerr << "Fatal error: " << e.what() << std::endl;
        return 1;
    }

    return 0;
}