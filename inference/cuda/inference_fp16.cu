#include "hip/hip_runtime.h"
#include </usr/include/hipDNN.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <cassert>
#include <algorithm>
#include <chrono>
#include <iomanip>
#include <hip/hip_fp16.h>
#include <sstream>
#include <algorithm>

// Error checking macros
#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)
template<typename T>
void check(T err, const char* const func, const char* const file,
           const int line) {
    if (err != hipSuccess) {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                  << std::endl;
        std::cerr << hipGetErrorString(err) << " " << func << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

template <typename T>
std::vector<T> loadBinaryFile(const std::string& filename) {
    std::ifstream file(filename, std::ios::binary);
    if (!file.is_open()) {
        std::cerr << "Error: Could not open file " << filename << std::endl;
        std::exit(EXIT_FAILURE);
    }
    
    file.seekg(0, std::ios::end);
    size_t file_size = file.tellg();
    file.seekg(0, std::ios::beg);
    
    size_t num_elements = file_size / sizeof(T);
    
    std::cout << "Loading " << filename << " - File size: " << file_size 
              << " bytes, Elements: " << num_elements << std::endl;
    
    std::vector<T> buffer(num_elements);
    file.read(reinterpret_cast<char*>(buffer.data()), file_size);
    file.close();
    
    return buffer;
}

#define CUDNN_CHECK(call)                                                         \
    {                                                                             \
        hipdnnStatus_t err = call;                                                 \
        if (err != HIPDNN_STATUS_SUCCESS) {                                        \
            std::cerr << "CuDNN Error: " << hipdnnGetErrorString(err) << std::endl; \
            std::exit(EXIT_FAILURE);                                              \
        }                                                                         \
    }

#define CUDA_CHECK(call)                                                         \
    {                                                                            \
        hipError_t err = call;                                                  \
        if (err != hipSuccess) {                                                \
            std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl; \
            std::exit(EXIT_FAILURE);                                             \
        }                                                                        \
    }

__global__ void floatToHalf(float* input, half* output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = __float2half(input[idx]);
    }
}

__global__ void halfToFloat(half* input, float* output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = __half2float(input[idx]);
    }
}

class CUDACNNInference {
public:
    CUDACNNInference(int batch_size, const std::string& weights_path);
    ~CUDACNNInference();
    void loadWeights(const std::string& weights_path);
    void initializeLayers();
    void infer(const std::vector<float>& input_data);
    std::vector<float> getOutput();

private:
    hipdnnHandle_t cudnn;
    hipdnnTensorDescriptor_t input_desc;
    hipdnnTensorDescriptor_t conv1_output_desc;
    hipdnnTensorDescriptor_t pool1_output_desc;
    hipdnnTensorDescriptor_t conv2_output_desc;
    hipdnnTensorDescriptor_t pool2_output_desc;
    hipdnnTensorDescriptor_t pool2_flat_desc;
    hipdnnTensorDescriptor_t fc1_input_desc;
    hipdnnTensorDescriptor_t fc1_output_desc;
    hipdnnTensorDescriptor_t fc2_input_desc;
    hipdnnTensorDescriptor_t fc2_output_desc;
    hipdnnFilterDescriptor_t fc1_weight_desc;
    hipdnnFilterDescriptor_t fc2_weight_desc;
    
    hipdnnTensorDescriptor_t conv1_bias_desc;
    hipdnnTensorDescriptor_t conv2_bias_desc;
    hipdnnTensorDescriptor_t fc1_bias_desc;
    hipdnnTensorDescriptor_t fc2_bias_desc;
    
    hipdnnFilterDescriptor_t conv1_filter_desc;
    hipdnnFilterDescriptor_t conv2_filter_desc;
    hipdnnConvolutionDescriptor_t conv1_desc;
    hipdnnConvolutionDescriptor_t conv2_desc;
    hipdnnConvolutionDescriptor_t fc1_desc;
    hipdnnConvolutionDescriptor_t fc2_desc;
    
    hipdnnActivationDescriptor_t relu_activation;
    hipdnnPoolingDescriptor_t pooling_desc;

    int fc1_input_size;

    float *d_input;
    half *d_conv1_weight, *d_conv1_bias, *d_conv1_output;
    half *d_pool1_output;
    half *d_conv2_weight, *d_conv2_bias, *d_conv2_output;
    half *d_pool2_output;
    half *d_fc1_weight, *d_fc1_bias, *d_fc1_output;
    half *d_fc2_weight, *d_fc2_bias, *d_fc2_output;

    size_t workspace_size;
    void *d_workspace;

    int batch_size;

    struct LayerDims {
        int n, c, h, w;
    };
    LayerDims conv1_dims, pool1_dims, conv2_dims, pool2_dims, fc1_dims, fc2_dims;
};

CUDACNNInference::CUDACNNInference(int batch_size_, const std::string& weights_path)
    : batch_size(batch_size_) {
    std::cout << "Initializing CuDNN..." << std::endl;
    CUDNN_CHECK(hipdnnCreate(&cudnn));

    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&input_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&conv1_output_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&pool1_output_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&conv2_output_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&pool2_output_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&pool2_flat_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&fc1_input_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&fc1_output_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&fc2_input_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&fc2_output_desc));
    CUDNN_CHECK(hipdnnCreateFilterDescriptor(&fc1_weight_desc));
    CUDNN_CHECK(hipdnnCreateFilterDescriptor(&fc2_weight_desc));

    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&conv1_bias_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&conv2_bias_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&fc1_bias_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&fc2_bias_desc));
    
    CUDNN_CHECK(hipdnnCreateFilterDescriptor(&conv1_filter_desc));
    CUDNN_CHECK(hipdnnCreateFilterDescriptor(&conv2_filter_desc));
    CUDNN_CHECK(hipdnnCreateConvolutionDescriptor(&conv1_desc));
    CUDNN_CHECK(hipdnnCreateConvolutionDescriptor(&conv2_desc));
    CUDNN_CHECK(hipdnnCreateConvolutionDescriptor(&fc1_desc));
    CUDNN_CHECK(hipdnnCreateConvolutionDescriptor(&fc2_desc));
    
    CUDNN_CHECK(hipdnnCreateActivationDescriptor(&relu_activation));
    CUDNN_CHECK(hipdnnCreatePoolingDescriptor(&pooling_desc));

    loadWeights(weights_path);
    initializeLayers();
}

void CUDACNNInference::loadWeights(const std::string& weights_path) {
    std::cout << "Loading FP16 model weights..." << std::endl;
    
    // Build full paths to the weight files
    std::string conv1_weight_path = weights_path + "/conv1.weight_fp16.bin";
    std::string conv1_bias_path = weights_path + "/conv1.bias_fp16.bin";
    std::string conv2_weight_path = weights_path + "/conv2.weight_fp16.bin";
    std::string conv2_bias_path = weights_path + "/conv2.bias_fp16.bin";
    std::string fc1_weight_path = weights_path + "/fc1.weight_fp16.bin";
    std::string fc1_bias_path = weights_path + "/fc1.bias_fp16.bin";
    std::string fc2_weight_path = weights_path + "/fc2.weight_fp16.bin";
    std::string fc2_bias_path = weights_path + "/fc2.bias_fp16.bin";

    // Load weights and biases
    auto conv1_weights = loadBinaryFile<half>(conv1_weight_path);
    auto conv1_biases = loadBinaryFile<half>(conv1_bias_path);
    auto conv2_weights = loadBinaryFile<half>(conv2_weight_path);
    auto conv2_biases = loadBinaryFile<half>(conv2_bias_path);
    auto fc1_weights = loadBinaryFile<half>(fc1_weight_path);
    auto fc1_biases = loadBinaryFile<half>(fc1_bias_path);
    auto fc2_weights = loadBinaryFile<half>(fc2_weight_path);
    auto fc2_biases = loadBinaryFile<half>(fc2_bias_path);

    // Verify sizes based on model architecture
    const size_t conv1_weights_size = 32 * 3 * 3 * 3;
    const size_t conv1_bias_size = 32;
    const size_t conv2_weights_size = 64 * 32 * 3 * 3;
    const size_t conv2_bias_size = 64;
    const size_t fc1_weights_size = 128 * (64 * 8 * 8);
    const size_t fc1_bias_size = 128;
    const size_t fc2_weights_size = 10 * 128;
    const size_t fc2_bias_size = 10;
    
    // Allocate and copy weights to device
    CUDA_CHECK(hipMalloc(&d_conv1_weight, conv1_weights_size * sizeof(half)));
    CUDA_CHECK(hipMalloc(&d_conv1_bias, conv1_bias_size * sizeof(half)));
    CUDA_CHECK(hipMemcpy(d_conv1_weight, conv1_weights.data(), 
                         conv1_weights_size * sizeof(half), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_conv1_bias, conv1_biases.data(), 
                         conv1_bias_size * sizeof(half), hipMemcpyHostToDevice));
    
    CUDA_CHECK(hipMalloc(&d_conv2_weight, conv2_weights_size * sizeof(half)));
    CUDA_CHECK(hipMalloc(&d_conv2_bias, conv2_bias_size * sizeof(half)));
    CUDA_CHECK(hipMemcpy(d_conv2_weight, conv2_weights.data(),
                         conv2_weights_size * sizeof(half), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_conv2_bias, conv2_biases.data(),
                         conv2_bias_size * sizeof(half), hipMemcpyHostToDevice));

    CUDA_CHECK(hipMalloc(&d_fc1_weight, fc1_weights_size * sizeof(half)));
    CUDA_CHECK(hipMalloc(&d_fc1_bias, fc1_bias_size * sizeof(half)));
    CUDA_CHECK(hipMemcpy(d_fc1_weight, fc1_weights.data(),
                         fc1_weights_size * sizeof(half), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_fc1_bias, fc1_biases.data(),
                         fc1_bias_size * sizeof(half), hipMemcpyHostToDevice));

    CUDA_CHECK(hipMalloc(&d_fc2_weight, fc2_weights_size * sizeof(half)));
    CUDA_CHECK(hipMalloc(&d_fc2_bias, fc2_bias_size * sizeof(half)));
    CUDA_CHECK(hipMemcpy(d_fc2_weight, fc2_weights.data(),
                         fc2_weights_size * sizeof(half), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_fc2_bias, fc2_biases.data(),
                         fc2_bias_size * sizeof(half), hipMemcpyHostToDevice));

    std::cout << "Successfully loaded all weights to GPU." << std::endl;
}

void CUDACNNInference::initializeLayers() {
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(input_desc, HIPDNN_TENSOR_NCHW, 
        HIPDNN_DATA_HALF, batch_size, 3, 32, 32));

    CUDNN_CHECK(hipdnnSetFilter4dDescriptor(conv1_filter_desc, HIPDNN_DATA_HALF, 
        HIPDNN_TENSOR_NCHW, 32, 3, 3, 3));
    
    CUDNN_CHECK(hipdnnSetConvolution2dDescriptor(conv1_desc, 
        1, 1,    // padding
        1, 1,    // stride
        1, 1,    // dilation
        HIPDNN_CROSS_CORRELATION, 
        HIPDNN_DATA_HALF));
    CUDNN_CHECK(hipdnnSetConvolutionMathType(conv1_desc, HIPDNN_DEFAULT_MATH));

    CUDNN_CHECK(hipdnnGetConvolution2dForwardOutputDim(conv1_desc, input_desc, 
        conv1_filter_desc, &conv1_dims.n, &conv1_dims.c, &conv1_dims.h, &conv1_dims.w));
    
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(conv1_output_desc, HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_HALF, conv1_dims.n, conv1_dims.c, conv1_dims.h, conv1_dims.w));
    
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(conv1_bias_desc, HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_HALF, 1, conv1_dims.c, 1, 1));

    CUDNN_CHECK(hipdnnSetPooling2dDescriptor(pooling_desc, 
        HIPDNN_POOLING_MAX,
        HIPDNN_NOT_PROPAGATE_NAN,
        2, 2,    // window size
        0, 0,    // padding
        2, 2));  // stride

    CUDNN_CHECK(hipdnnGetPooling2dForwardOutputDim(pooling_desc,
        conv1_output_desc,
        &pool1_dims.n, &pool1_dims.c, &pool1_dims.h, &pool1_dims.w));
    
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(pool1_output_desc, HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_HALF, pool1_dims.n, pool1_dims.c, pool1_dims.h, pool1_dims.w));

    CUDNN_CHECK(hipdnnSetFilter4dDescriptor(conv2_filter_desc, HIPDNN_DATA_HALF, 
        HIPDNN_TENSOR_NCHW, 64, 32, 3, 3));
    
    CUDNN_CHECK(hipdnnSetConvolution2dDescriptor(conv2_desc, 
        1, 1,    // padding
        1, 1,    // stride
        1, 1,    // dilation
        HIPDNN_CROSS_CORRELATION, 
        HIPDNN_DATA_HALF));
    CUDNN_CHECK(hipdnnSetConvolutionMathType(conv2_desc, HIPDNN_DEFAULT_MATH));
    
    CUDNN_CHECK(hipdnnGetConvolution2dForwardOutputDim(conv2_desc, pool1_output_desc, 
        conv2_filter_desc, &conv2_dims.n, &conv2_dims.c, &conv2_dims.h, &conv2_dims.w));
    
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(conv2_output_desc, HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_HALF, conv2_dims.n, conv2_dims.c, conv2_dims.h, conv2_dims.w));
    
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(conv2_bias_desc, HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_HALF, 1, conv2_dims.c, 1, 1));

    pool2_dims = {conv2_dims.n, conv2_dims.c, conv2_dims.h/2, conv2_dims.w/2};
    
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(pool2_output_desc, HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_HALF, pool2_dims.n, pool2_dims.c, pool2_dims.h, pool2_dims.w));

    fc1_input_size = pool2_dims.c * pool2_dims.h * pool2_dims.w;

    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(pool2_flat_desc, HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_HALF, batch_size, fc1_input_size, 1, 1));

    CUDNN_CHECK(hipdnnSetFilter4dDescriptor(fc1_weight_desc, HIPDNN_DATA_HALF,
        HIPDNN_TENSOR_NCHW, 128, fc1_input_size, 1, 1));
    
    CUDNN_CHECK(hipdnnSetConvolution2dDescriptor(fc1_desc,
        0, 0,    // padding
        1, 1,    // stride
        1, 1,    // dilation
        HIPDNN_CROSS_CORRELATION,
        HIPDNN_DATA_HALF));
    CUDNN_CHECK(hipdnnSetConvolutionMathType(fc1_desc, HIPDNN_DEFAULT_MATH));

    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(fc1_output_desc, HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_HALF, batch_size, 128, 1, 1));
    
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(fc1_bias_desc, HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_HALF, 1, 128, 1, 1));

    CUDNN_CHECK(hipdnnSetFilter4dDescriptor(fc2_weight_desc, HIPDNN_DATA_HALF,
        HIPDNN_TENSOR_NCHW, 10, 128, 1, 1));
    
    CUDNN_CHECK(hipdnnSetConvolution2dDescriptor(fc2_desc,
        0, 0,    // padding
        1, 1,    // stride
        1, 1,    // dilation
        HIPDNN_CROSS_CORRELATION,
        HIPDNN_DATA_HALF));
    CUDNN_CHECK(hipdnnSetConvolutionMathType(fc2_desc, HIPDNN_DEFAULT_MATH));

    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(fc2_output_desc, HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_HALF, batch_size, 10, 1, 1));
    
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(fc2_bias_desc, HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_HALF, 1, 10, 1, 1));

    CUDNN_CHECK(hipdnnSetActivationDescriptor(relu_activation,
        HIPDNN_ACTIVATION_RELU,
        HIPDNN_NOT_PROPAGATE_NAN,
        0.0));

    size_t workspace_sizes[4];
    CUDNN_CHECK(hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
        input_desc, conv1_filter_desc, conv1_desc, conv1_output_desc,
        HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM, &workspace_sizes[0]));
    
    CUDNN_CHECK(hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
        pool1_output_desc, conv2_filter_desc, conv2_desc, conv2_output_desc,
        HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM, &workspace_sizes[1]));
    
    CUDNN_CHECK(hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
        pool2_flat_desc, fc1_weight_desc, fc1_desc, fc1_output_desc,
        HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM, &workspace_sizes[2]));
    
    CUDNN_CHECK(hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
        fc1_output_desc, fc2_weight_desc, fc2_desc, fc2_output_desc,
        HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM, &workspace_sizes[3]));

    workspace_size = *std::max_element(workspace_sizes, workspace_sizes + 4);
    CUDA_CHECK(hipMalloc(&d_workspace, workspace_size));

    size_t input_bytes = batch_size * 3 * 32 * 32 * sizeof(float);
    size_t conv1_output_bytes = batch_size * conv1_dims.c * conv1_dims.h * conv1_dims.w * sizeof(half);
    size_t pool1_output_bytes = batch_size * pool1_dims.c * pool1_dims.h * pool1_dims.w * sizeof(half);
    size_t conv2_output_bytes = batch_size * conv2_dims.c * conv2_dims.h * conv2_dims.w * sizeof(half);
    size_t pool2_output_bytes = batch_size * pool2_dims.c * pool2_dims.h * pool2_dims.w * sizeof(half);
    size_t fc1_output_bytes = batch_size * 128 * sizeof(half);
    size_t fc2_output_bytes = batch_size * 10 * sizeof(half);

    CUDA_CHECK(hipMalloc(&d_input, input_bytes));
    CUDA_CHECK(hipMalloc(&d_conv1_output, conv1_output_bytes));
    CUDA_CHECK(hipMalloc(&d_pool1_output, pool1_output_bytes));
    CUDA_CHECK(hipMalloc(&d_conv2_output, conv2_output_bytes));
    CUDA_CHECK(hipMalloc(&d_pool2_output, pool2_output_bytes));
    CUDA_CHECK(hipMalloc(&d_fc1_output, fc1_output_bytes));
    CUDA_CHECK(hipMalloc(&d_fc2_output, fc2_output_bytes));

    std::cout << "Layer initialization complete." << std::endl;
    std::cout << "Workspace size: " << workspace_size << " bytes" << std::endl;
}

void CUDACNNInference::infer(const std::vector<float>& input_data) {
    const float alpha_f = 1.0f;
    const float beta_f = 0.0f;
    const void* alpha_ptr = &alpha_f;
    const void* beta_ptr = &beta_f;
    
    size_t expected_input_size = batch_size * 3 * 32 * 32;
    if (input_data.size() != expected_input_size) {
        throw std::runtime_error("Input data size mismatch");
    }

    float* d_input_float;
    CUDA_CHECK(hipMalloc(&d_input_float, input_data.size() * sizeof(float)));
    CUDA_CHECK(hipMemcpy(d_input_float, input_data.data(), input_data.size() * sizeof(float), hipMemcpyHostToDevice));

    int blockSize = 256;
    int numBlocks = (input_data.size() + blockSize - 1) / blockSize;
    floatToHalf<<<numBlocks, blockSize>>>(d_input_float, (half*)d_input, input_data.size());
    CHECK_CUDA_ERROR(hipGetLastError());
    CUDA_CHECK(hipFree(d_input_float));

    CUDNN_CHECK(hipdnnConvolutionForward(cudnn, alpha_ptr, 
        input_desc, d_input,
        conv1_filter_desc, d_conv1_weight,
        conv1_desc, HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM,
        d_workspace, workspace_size,
        beta_ptr, conv1_output_desc, d_conv1_output));
    
    CUDNN_CHECK(hipdnnAddTensor(cudnn, 
        alpha_ptr,
        conv1_bias_desc, d_conv1_bias,
        alpha_ptr,
        conv1_output_desc, d_conv1_output));
    
    CUDNN_CHECK(hipdnnActivationForward(cudnn, relu_activation,
        alpha_ptr, conv1_output_desc, d_conv1_output,
        beta_ptr, conv1_output_desc, d_conv1_output));

    CUDNN_CHECK(hipdnnPoolingForward(cudnn, pooling_desc,
        alpha_ptr, conv1_output_desc, d_conv1_output,
        beta_ptr, pool1_output_desc, d_pool1_output));

    CUDNN_CHECK(hipdnnConvolutionForward(cudnn, alpha_ptr,
        pool1_output_desc, d_pool1_output,
        conv2_filter_desc, d_conv2_weight,
        conv2_desc, HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM,
        d_workspace, workspace_size,
        beta_ptr, conv2_output_desc, d_conv2_output));
    
    CUDNN_CHECK(hipdnnAddTensor(cudnn, 
        alpha_ptr,
        conv2_bias_desc, d_conv2_bias,
        alpha_ptr,
        conv2_output_desc, d_conv2_output));
    
    CUDNN_CHECK(hipdnnActivationForward(cudnn, relu_activation,
        alpha_ptr, conv2_output_desc, d_conv2_output,
        beta_ptr, conv2_output_desc, d_conv2_output));

    CUDNN_CHECK(hipdnnPoolingForward(cudnn, pooling_desc,
        alpha_ptr, conv2_output_desc, d_conv2_output,
        beta_ptr, pool2_output_desc, d_pool2_output));

    CUDNN_CHECK(hipdnnConvolutionForward(cudnn, alpha_ptr,
        pool2_flat_desc, d_pool2_output,
        fc1_weight_desc, d_fc1_weight,
        fc1_desc, HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
        d_workspace, workspace_size,
        beta_ptr, fc1_output_desc, d_fc1_output));
    
    CUDNN_CHECK(hipdnnAddTensor(cudnn, 
        alpha_ptr,
        fc1_bias_desc, d_fc1_bias,
        alpha_ptr,
        fc1_output_desc, d_fc1_output));
    
    CUDNN_CHECK(hipdnnActivationForward(cudnn, relu_activation,
        alpha_ptr, fc1_output_desc, d_fc1_output,
        beta_ptr, fc1_output_desc, d_fc1_output));

    CUDNN_CHECK(hipdnnConvolutionForward(cudnn, alpha_ptr,
        fc1_output_desc, d_fc1_output,
        fc2_weight_desc, d_fc2_weight,
        fc2_desc, HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
        d_workspace, workspace_size,
        beta_ptr, fc2_output_desc, d_fc2_output));
    
    CUDNN_CHECK(hipdnnAddTensor(cudnn, 
        alpha_ptr,
        fc2_bias_desc, d_fc2_bias,
        alpha_ptr,
        fc2_output_desc, d_fc2_output));

    CHECK_CUDA_ERROR(hipGetLastError());
}

std::vector<float> CUDACNNInference::getOutput() {
    size_t output_size = batch_size * 10;
    std::vector<float> output(output_size);
    
    float* d_output_float;
    CUDA_CHECK(hipMalloc(&d_output_float, output_size * sizeof(float)));
    
    int blockSize = 256;
    int numBlocks = (output_size + blockSize - 1) / blockSize;
    halfToFloat<<<numBlocks, blockSize>>>((half*)d_fc2_output, d_output_float, output_size);
    CHECK_CUDA_ERROR(hipGetLastError());
    
    CUDA_CHECK(hipMemcpy(output.data(), d_output_float, output.size() * sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipFree(d_output_float));
    
    // Apply softmax normalization for each sample
    for (int i = 0; i < batch_size; ++i) {
        float max_val = *std::max_element(output.begin() + i * 10, output.begin() + (i + 1) * 10);
        float sum = 0.0f;

        for (int j = 0; j < 10; ++j) {
            output[i * 10 + j] = std::exp(output[i * 10 + j] - max_val);
            sum += output[i * 10 + j];
        }

        for (int j = 0; j < 10; ++j) {
            output[i * 10 + j] /= sum;
        }
    }
    
    return output;
}

CUDACNNInference::~CUDACNNInference() {
    hipFree(d_input);
    hipFree(d_conv1_weight);
    hipFree(d_conv1_bias);
    hipFree(d_conv1_output);
    hipFree(d_pool1_output);
    hipFree(d_conv2_weight);
    hipFree(d_conv2_bias);
    hipFree(d_conv2_output);
    hipFree(d_pool2_output);
    hipFree(d_fc1_weight);
    hipFree(d_fc1_bias);
    hipFree(d_fc1_output);
    hipFree(d_fc2_weight);
    hipFree(d_fc2_bias);
    hipFree(d_fc2_output);
    hipFree(d_workspace);

    hipdnnDestroyTensorDescriptor(input_desc);
    hipdnnDestroyTensorDescriptor(conv1_output_desc);
    hipdnnDestroyTensorDescriptor(pool1_output_desc);
    hipdnnDestroyTensorDescriptor(conv2_output_desc);
    hipdnnDestroyTensorDescriptor(pool2_output_desc);
    hipdnnDestroyTensorDescriptor(pool2_flat_desc);
    hipdnnDestroyTensorDescriptor(fc1_input_desc);
    hipdnnDestroyTensorDescriptor(fc1_output_desc);
    hipdnnDestroyTensorDescriptor(fc2_input_desc);
    hipdnnDestroyTensorDescriptor(fc2_output_desc);
    hipdnnDestroyFilterDescriptor(fc1_weight_desc);
    hipdnnDestroyFilterDescriptor(fc2_weight_desc);

    hipdnnDestroyTensorDescriptor(conv1_bias_desc);
    hipdnnDestroyTensorDescriptor(conv2_bias_desc);
    hipdnnDestroyTensorDescriptor(fc1_bias_desc);
    hipdnnDestroyTensorDescriptor(fc2_bias_desc);

    hipdnnDestroyFilterDescriptor(conv1_filter_desc);
    hipdnnDestroyFilterDescriptor(conv2_filter_desc);
    hipdnnDestroyConvolutionDescriptor(conv1_desc);
    hipdnnDestroyConvolutionDescriptor(conv2_desc);
    hipdnnDestroyConvolutionDescriptor(fc1_desc);
    hipdnnDestroyConvolutionDescriptor(fc2_desc);

    hipdnnDestroyActivationDescriptor(relu_activation);
    hipdnnDestroyPoolingDescriptor(pooling_desc);

    hipdnnDestroy(cudnn);
}

void parseArguments(int argc, char** argv, int& gpu_id, int& repeat_factor, size_t& batch_size, std::string& data_path, std::string& weights_path) {
    if (argc >= 6) {
        gpu_id = std::atoi(argv[1]);
        repeat_factor = std::atoi(argv[2]);
        batch_size = std::stoul(argv[3]);  // Use std::stoul for size_t
        data_path = argv[4];
        weights_path = argv[5];
    } else {
        std::cerr << "Usage: " << argv[0] << " <gpu_id> <repeat_factor> <batch_size> <data_path> <weights_path>" << std::endl;
        std::exit(EXIT_FAILURE);
    }
}


int main(int argc, char** argv) {
    int gpu_id = 0;
    int repeat_factor = 1;
    size_t batch_size = 256;// Default value
    std::string data_path;
    std::string weights_path;

    parseArguments(argc, argv, gpu_id, repeat_factor, batch_size, data_path, weights_path);
    CUDA_CHECK(hipSetDevice(gpu_id));

    std::cout << "Running on GPU: " << gpu_id << std::endl;
    std::cout << "Repeat factor: " << repeat_factor << std::endl;
    std::cout << "Batch size: " << batch_size << std::endl;

    try {
        std::cout << "Loading validation data..." << std::endl;

        // Build paths to validation data files
        std::string validation_images_path = data_path + "/validation_images.bin";
        std::string validation_labels_path = data_path + "/validation_labels.bin";

        auto validation_images = loadBinaryFile<float>(validation_images_path);
        auto validation_labels = loadBinaryFile<int>(validation_labels_path);

        size_t image_size = 3 * 32 * 32;
        std::vector<std::vector<float>> images;
        for (size_t i = 0; i < validation_images.size(); i += image_size) {
            images.emplace_back(validation_images.begin() + i,
                                validation_images.begin() + i + image_size);
        }

        // Repeat images and labels
        std::vector<std::vector<float>> repeated_images;
        std::vector<int> repeated_labels;

        for (int i = 0; i < repeat_factor; ++i) {
            repeated_images.insert(repeated_images.end(), images.begin(), images.end());
            repeated_labels.insert(repeated_labels.end(), validation_labels.begin(), validation_labels.end());
        }

        size_t total_images = repeated_images.size();
        std::cout << "Total images after repeating: " << total_images << std::endl;

        // Create CNN object with batch_size and weights_path
        CUDACNNInference cnn(batch_size, weights_path);

        std::cout << "\n=== Starting Evaluation ===" << std::endl;
        std::cout << "Model type: CUDA FP16" << std::endl;

        hipEvent_t start, stop;
        CUDA_CHECK(hipEventCreate(&start));
        CUDA_CHECK(hipEventCreate(&stop));

        size_t correct_count = 0;
        float total_time = 0.0f;

        // Warmup run with first batch
        std::cout << "Performing warmup runs..." << std::endl;
        std::vector<float> warmup_batch;
        warmup_batch.reserve(batch_size * image_size);
        for (int i = 0; i < batch_size && i < total_images; ++i) {
            warmup_batch.insert(warmup_batch.end(), repeated_images[i].begin(), repeated_images[i].end());
        }
        for (int i = 0; i < 10; i++) {
            cnn.infer(warmup_batch);
        }

        // Main evaluation loop with batching
        std::cout << "Starting main evaluation..." << std::endl;
        size_t total_batches = (total_images + batch_size - 1) / batch_size;
        
        for (size_t batch_idx = 0; batch_idx < total_batches; ++batch_idx) {
            size_t batch_start = batch_idx * batch_size;
            size_t batch_end = std::min(batch_start + batch_size, total_images);
            size_t current_batch_size = batch_end - batch_start;

            std::vector<float> batch_input;
            batch_input.reserve(batch_size * image_size);

            // Load actual images
            for (size_t i = batch_start; i < batch_end; ++i) {
                batch_input.insert(batch_input.end(), repeated_images[i].begin(), repeated_images[i].end());
            }

            // Pad the batch if necessary
            if (current_batch_size < batch_size) {
                // Duplicate the last image to fill the batch
                const auto& last_image = repeated_images[batch_end - 1];
                for (size_t i = current_batch_size; i < batch_size; ++i) {
                    batch_input.insert(batch_input.end(), last_image.begin(), last_image.end());
                }
            }
            CUDA_CHECK(hipEventRecord(start));
            cnn.infer(batch_input);
            std::vector<float> output = cnn.getOutput();
            CUDA_CHECK(hipEventRecord(stop));
            CUDA_CHECK(hipEventSynchronize(stop));
            
            float milliseconds = 0;
            CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
            total_time += milliseconds;

            for (size_t i = 0; i < current_batch_size; ++i) {
                int predicted_label = std::distance(
                    output.begin() + i * 10,
                    std::max_element(output.begin() + i * 10, output.begin() + (i + 1) * 10)
                );
                if (predicted_label == repeated_labels[batch_start + i]) {
                    ++correct_count;
                }
            }

            if (batch_idx % 10 == 0) {
                float running_accuracy = (static_cast<float>(correct_count) / ((batch_idx + 1) * batch_size)) * 100.0f;
                std::cout << "Processed " << (batch_idx + 1) * batch_size << "/" << total_images 
                         << " images. Running accuracy: " << std::fixed 
                         << std::setprecision(2) << running_accuracy << "%" << std::endl;
            }
        }

        float accuracy = static_cast<float>(correct_count) / total_images * 100.0f;
        float avg_time = total_time / total_batches;  // Average time per batch
        float throughput = (batch_size * 1000.0f) / avg_time;  // Images per second

        std::cout << "\n=== Final Results ===" << std::endl;
        std::cout << "Model type: CUDA FP16" << std::endl;
        std::cout << "Batch size: " << batch_size << std::endl;
        std::cout << "Total images: " << total_images << std::endl;
        std::cout << "Correct predictions: " << correct_count << std::endl;
        std::cout << "Accuracy: " << std::fixed << std::setprecision(2) << accuracy << "%" << std::endl;
        std::cout << "Average batch inference time: " << std::fixed << std::setprecision(3) 
                 << avg_time << " ms" << std::endl;
        std::cout << "Throughput: " << std::fixed << std::setprecision(1) 
                 << throughput << " images/second" << std::endl;
        std::cout << "Total evaluation time: " << std::fixed << std::setprecision(2) 
                 << total_time / 1000.0f << " seconds" << std::endl;

        CUDA_CHECK(hipEventDestroy(start));
        CUDA_CHECK(hipEventDestroy(stop));

    } catch (const std::exception& e) {
        std::cerr << "Fatal error: " << e.what() << std::endl;
        return 1;
    }

    return 0;
}
