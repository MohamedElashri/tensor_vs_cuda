#include "hip/hip_runtime.h"
// CUDA Cores Inference Engine


#include </usr/include/hipDNN.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <cassert>
#include <algorithm>
#include <chrono>
#include <iomanip>

// Load binary data from file
template <typename T>
std::vector<T> loadBinaryFile(const std::string& filename) {
    std::ifstream file(filename, std::ios::binary);
    if (!file.is_open()) {
        std::cerr << "Error: Could not open file " << filename << std::endl;
        std::exit(EXIT_FAILURE);
    }
    file.seekg(0, std::ios::end);
    size_t size = file.tellg() / sizeof(T);
    file.seekg(0, std::ios::beg);
    std::vector<T> buffer(size);
    file.read(reinterpret_cast<char*>(buffer.data()), size * sizeof(T));
    file.close();
    return buffer;
}

// Helper function to handle CuDNN errors
#define CUDNN_CHECK(call)                                                         \
    {                                                                             \
        hipdnnStatus_t err = call;                                                 \
        if (err != HIPDNN_STATUS_SUCCESS) {                                        \
            std::cerr << "CuDNN Error: " << hipdnnGetErrorString(err) << std::endl; \
            std::exit(EXIT_FAILURE);                                              \
        }                                                                         \
    }

// Helper function to handle CUDA errors

#define CUDA_CHECK(call)                                                         \
    {                                                                            \
        hipError_t err = call;                                                  \
        if (err != hipSuccess) {                                                \
            std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl; \
            std::exit(EXIT_FAILURE);                                             \
        }                                                                        \
    }

class CUDACNNInference {
public:
    CUDACNNInference();
    ~CUDACNNInference();
    void loadWeights();
    void initializeLayers();
    void infer(const std::vector<float>& input_data);
    std::vector<float> getOutput();
    void evaluate(const std::vector<std::vector<float>>& images, const std::vector<int>& labels);

private:
    hipdnnHandle_t cudnn;
    
    // Layer descriptors
    hipdnnTensorDescriptor_t input_desc;
    hipdnnTensorDescriptor_t conv1_output_desc;
    hipdnnTensorDescriptor_t pool1_output_desc;
    hipdnnTensorDescriptor_t conv2_output_desc;
    hipdnnTensorDescriptor_t pool2_output_desc;
    hipdnnTensorDescriptor_t pool2_flat_desc;
    hipdnnTensorDescriptor_t fc1_input_desc;
    hipdnnTensorDescriptor_t fc1_output_desc;
    hipdnnTensorDescriptor_t fc2_input_desc;
    hipdnnTensorDescriptor_t fc2_output_desc;
    hipdnnFilterDescriptor_t fc1_weight_desc;
    hipdnnFilterDescriptor_t fc2_weight_desc;
    
    // Bias descriptors
    hipdnnTensorDescriptor_t conv1_bias_desc;
    hipdnnTensorDescriptor_t conv2_bias_desc;
    hipdnnTensorDescriptor_t fc1_bias_desc;
    hipdnnTensorDescriptor_t fc2_bias_desc;
    
    // Convolution descriptors
    hipdnnFilterDescriptor_t conv1_filter_desc;
    hipdnnFilterDescriptor_t conv2_filter_desc;
    hipdnnConvolutionDescriptor_t conv1_desc;
    hipdnnConvolutionDescriptor_t conv2_desc;
    
    // FC layer convolution descriptors
    hipdnnConvolutionDescriptor_t fc1_desc;
    hipdnnConvolutionDescriptor_t fc2_desc;
    
    // Activation and pooling descriptors
    hipdnnActivationDescriptor_t relu_activation;
    hipdnnPoolingDescriptor_t pooling_desc;

    int fc1_input_size;


    // Device memory pointers
    float *d_input;
    float *d_conv1_weight, *d_conv1_bias, *d_conv1_output;
    float *d_pool1_output;
    float *d_conv2_weight, *d_conv2_bias, *d_conv2_output;
    float *d_pool2_output;
    float *d_fc1_weight, *d_fc1_bias, *d_fc1_output;
    float *d_fc2_weight, *d_fc2_bias, *d_fc2_output;

    // Workspace for convolutions
    size_t workspace_size;
    void *d_workspace;

    // Output dimensions
    int batch_size = 1;
    struct LayerDims {
        int n, c, h, w;
    };
    LayerDims conv1_dims, pool1_dims, conv2_dims, pool2_dims, fc1_dims, fc2_dims;
};

CUDACNNInference::CUDACNNInference() {
    std::cout << "Initializing CuDNN..." << std::endl;
    CUDNN_CHECK(hipdnnCreate(&cudnn));

    // Create all descriptors
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&input_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&conv1_output_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&pool1_output_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&conv2_output_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&pool2_output_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&pool2_flat_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&fc1_input_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&fc1_output_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&fc2_input_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&fc2_output_desc));
    CUDNN_CHECK(hipdnnCreateFilterDescriptor(&fc1_weight_desc));
    CUDNN_CHECK(hipdnnCreateFilterDescriptor(&fc2_weight_desc));

    // Create bias descriptors
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&conv1_bias_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&conv2_bias_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&fc1_bias_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&fc2_bias_desc));
    
    // Create filter and convolution descriptors
    CUDNN_CHECK(hipdnnCreateFilterDescriptor(&conv1_filter_desc));
    CUDNN_CHECK(hipdnnCreateFilterDescriptor(&conv2_filter_desc));
    CUDNN_CHECK(hipdnnCreateConvolutionDescriptor(&conv1_desc));
    CUDNN_CHECK(hipdnnCreateConvolutionDescriptor(&conv2_desc));
    CUDNN_CHECK(hipdnnCreateConvolutionDescriptor(&fc1_desc));
    CUDNN_CHECK(hipdnnCreateConvolutionDescriptor(&fc2_desc));
    
    CUDNN_CHECK(hipdnnCreateActivationDescriptor(&relu_activation));
    CUDNN_CHECK(hipdnnCreatePoolingDescriptor(&pooling_desc));

    loadWeights();
    initializeLayers();
}

CUDACNNInference::~CUDACNNInference() {
    // Free device memory for layer outputs
    hipFree(d_input);
    hipFree(d_conv1_output);
    hipFree(d_pool1_output);
    hipFree(d_conv2_output);
    hipFree(d_pool2_output);
    hipFree(d_fc1_output);
    hipFree(d_fc2_output);

    // Free device memory for weights and biases
    hipFree(d_conv1_weight);
    hipFree(d_conv1_bias);
    hipFree(d_conv2_weight);
    hipFree(d_conv2_bias);
    hipFree(d_fc1_weight);
    hipFree(d_fc1_bias);
    hipFree(d_fc2_weight);
    hipFree(d_fc2_bias);

    // Free workspace memory
    if (d_workspace) {
        hipFree(d_workspace);
    }

    // Destroy tensor descriptors
    hipdnnDestroyTensorDescriptor(input_desc);
    hipdnnDestroyTensorDescriptor(conv1_output_desc);
    hipdnnDestroyTensorDescriptor(pool1_output_desc);
    hipdnnDestroyTensorDescriptor(conv2_output_desc);
    hipdnnDestroyTensorDescriptor(pool2_output_desc);
    hipdnnDestroyTensorDescriptor(pool2_flat_desc);
    hipdnnDestroyTensorDescriptor(fc1_input_desc);
    hipdnnDestroyTensorDescriptor(fc1_output_desc);
    hipdnnDestroyTensorDescriptor(fc2_input_desc);
    hipdnnDestroyTensorDescriptor(fc2_output_desc);
    hipdnnDestroyFilterDescriptor(fc1_weight_desc);
    hipdnnDestroyFilterDescriptor(fc2_weight_desc);

    // Destroy bias descriptors
    hipdnnDestroyTensorDescriptor(conv1_bias_desc);
    hipdnnDestroyTensorDescriptor(conv2_bias_desc);
    hipdnnDestroyTensorDescriptor(fc1_bias_desc);
    hipdnnDestroyTensorDescriptor(fc2_bias_desc);

    // Destroy filter and convolution descriptors
    hipdnnDestroyFilterDescriptor(conv1_filter_desc);
    hipdnnDestroyFilterDescriptor(conv2_filter_desc);
    hipdnnDestroyConvolutionDescriptor(conv1_desc);
    hipdnnDestroyConvolutionDescriptor(conv2_desc);
    hipdnnDestroyConvolutionDescriptor(fc1_desc);
    hipdnnDestroyConvolutionDescriptor(fc2_desc);

    // Destroy activation and pooling descriptors
    hipdnnDestroyActivationDescriptor(relu_activation);
    hipdnnDestroyPoolingDescriptor(pooling_desc);

    // Destroy cuDNN handle
    hipdnnDestroy(cudnn);
}

void CUDACNNInference::loadWeights() {
    std::cout << "Loading model weights..." << std::endl;
    
    // Conv1 weights
    auto conv1_weights = loadBinaryFile<float>("../../../data/weights/conv1.weight_fp32.bin");
    auto conv1_biases = loadBinaryFile<float>("../../../data/weights/conv1.bias_fp32.bin");
    
    //  print first few weights for verification (debugging)
    // std::cout << "Conv1 weights first values: ";
    // for(int i = 0; i < 5; i++) {
    //     std::cout << conv1_weights[i] << " ";
    // }
    // std::cout << std::endl;
    
    const size_t conv1_weights_size = 32 * 3 * 3 * 3;
    const size_t conv1_bias_size = 32;
    
    if (conv1_weights.size() != conv1_weights_size || conv1_biases.size() != conv1_bias_size) {
        std::cerr << "Error: Conv1 weight/bias size mismatch!" << std::endl;
        std::exit(EXIT_FAILURE);
    }

    // Conv2 weights
    auto conv2_weights = loadBinaryFile<float>("../../../data/weights/conv2.weight_fp32.bin");
    auto conv2_biases = loadBinaryFile<float>("../../../data/weights/conv2.bias_fp32.bin");
    
    const size_t conv2_weights_size = 64 * 32 * 3 * 3;
    const size_t conv2_bias_size = 64;
    
    // FC1 weights
    auto fc1_weights = loadBinaryFile<float>("../../../data/weights/fc1.weight_fp32.bin");
    auto fc1_biases = loadBinaryFile<float>("../../../data/weights/fc1.bias_fp32.bin");
    
    const size_t fc1_weights_size = 128 * (64 * 8 * 8);
    const size_t fc1_bias_size = 128;  // Added definition
    
    // FC2 weights
    auto fc2_weights = loadBinaryFile<float>("../../../data/weights/fc2.weight_fp32.bin");
    auto fc2_biases = loadBinaryFile<float>("../../../data/weights/fc2.bias_fp32.bin");
    
    const size_t fc2_weights_size = 10 * 128;
    const size_t fc2_bias_size = 10;  // Added definition

    // Size verification
    if (conv2_weights.size() != conv2_weights_size || conv2_biases.size() != conv2_bias_size) {
        std::cerr << "Error: Conv2 weight/bias size mismatch!" << std::endl;
        std::exit(EXIT_FAILURE);
    }

    if (fc1_weights.size() != fc1_weights_size || fc1_biases.size() != fc1_bias_size) {
        std::cerr << "Error: FC1 weight/bias size mismatch!" << std::endl;
        std::exit(EXIT_FAILURE);
    }

    if (fc2_weights.size() != fc2_weights_size || fc2_biases.size() != fc2_bias_size) {
        std::cerr << "Error: FC2 weight/bias size mismatch!" << std::endl;
        std::exit(EXIT_FAILURE);
    }

    // Allocate and copy all weights in order
    hipMalloc(&d_conv1_weight, conv1_weights_size * sizeof(float));
    hipMalloc(&d_conv1_bias, conv1_bias_size * sizeof(float));
    hipMemcpy(d_conv1_weight, conv1_weights.data(), conv1_weights_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_conv1_bias, conv1_biases.data(), conv1_bias_size * sizeof(float), hipMemcpyHostToDevice);

    hipMalloc(&d_conv2_weight, conv2_weights_size * sizeof(float));
    hipMalloc(&d_conv2_bias, conv2_bias_size * sizeof(float));
    hipMemcpy(d_conv2_weight, conv2_weights.data(), conv2_weights_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_conv2_bias, conv2_biases.data(), conv2_bias_size * sizeof(float), hipMemcpyHostToDevice);

    hipMalloc(&d_fc1_weight, fc1_weights_size * sizeof(float));
    hipMalloc(&d_fc1_bias, fc1_bias_size * sizeof(float));
    hipMemcpy(d_fc1_weight, fc1_weights.data(), fc1_weights_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_fc1_bias, fc1_biases.data(), fc1_bias_size * sizeof(float), hipMemcpyHostToDevice);

    hipMalloc(&d_fc2_weight, fc2_weights_size * sizeof(float));
    hipMalloc(&d_fc2_bias, fc2_bias_size * sizeof(float));
    hipMemcpy(d_fc2_weight, fc2_weights.data(), fc2_weights_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_fc2_bias, fc2_biases.data(), fc2_bias_size * sizeof(float), hipMemcpyHostToDevice);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::cerr << "CUDA error while loading weights: " << hipGetErrorString(error) << std::endl;
        std::exit(EXIT_FAILURE);
    }
    
    std::cout << "Successfully loaded all weights to GPU." << std::endl;
}


void CUDACNNInference::initializeLayers() {
    // Input: 3x32x32
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(input_desc, HIPDNN_TENSOR_NCHW, 
        HIPDNN_DATA_FLOAT, batch_size, 3, 32, 32));

    // Conv1: 3 -> 32 channels, 3x3 kernel
    CUDNN_CHECK(hipdnnSetFilter4dDescriptor(conv1_filter_desc, HIPDNN_DATA_FLOAT, 
        HIPDNN_TENSOR_NCHW, 32, 3, 3, 3));
    
    CUDNN_CHECK(hipdnnSetConvolution2dDescriptor(conv1_desc, 
        1, 1,    // padding
        1, 1,    // stride
        1, 1,    // dilation
        HIPDNN_CROSS_CORRELATION, 
        HIPDNN_DATA_FLOAT));

    // Get Conv1 output dimensions
    CUDNN_CHECK(hipdnnGetConvolution2dForwardOutputDim(conv1_desc, input_desc, 
        conv1_filter_desc, &conv1_dims.n, &conv1_dims.c, &conv1_dims.h, &conv1_dims.w));
    
    std::cout << "Conv1 output dimensions: " << conv1_dims.n << "x" << conv1_dims.c 
              << "x" << conv1_dims.h << "x" << conv1_dims.w << std::endl;
    
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(conv1_output_desc, HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_FLOAT, conv1_dims.n, conv1_dims.c, conv1_dims.h, conv1_dims.w));
    
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(conv1_bias_desc, HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_FLOAT, 1, conv1_dims.c, 1, 1));

    // Find best algorithm for Conv1
    int requestedAlgoCount = 1;
    int returnedAlgoCount;
    hipdnnConvolutionFwdAlgoPerf_t perfResults;
    
    CUDNN_CHECK(cudnnGetConvolutionForwardAlgorithm_v7(cudnn,
        input_desc,
        conv1_filter_desc,
        conv1_desc,
        conv1_output_desc,
        requestedAlgoCount,
        &returnedAlgoCount,
        &perfResults));
    
    // Pooling setup
    CUDNN_CHECK(hipdnnSetPooling2dDescriptor(pooling_desc, 
        HIPDNN_POOLING_MAX,
        HIPDNN_NOT_PROPAGATE_NAN,
        2, 2,    // window height, width
        0, 0,    // padding height, width
        2, 2));  // stride height, width

    // Get Pool1 dimensions
    CUDNN_CHECK(hipdnnGetPooling2dForwardOutputDim(pooling_desc,
        conv1_output_desc,
        &pool1_dims.n, &pool1_dims.c, &pool1_dims.h, &pool1_dims.w));
    
    std::cout << "Pool1 dimensions: " << pool1_dims.n << "x" << pool1_dims.c 
              << "x" << pool1_dims.h << "x" << pool1_dims.w << std::endl;
    
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(pool1_output_desc, HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_FLOAT, pool1_dims.n, pool1_dims.c, pool1_dims.h, pool1_dims.w));

    // Conv2 setup
    CUDNN_CHECK(hipdnnSetFilter4dDescriptor(conv2_filter_desc, HIPDNN_DATA_FLOAT, 
        HIPDNN_TENSOR_NCHW, 64, 32, 3, 3));
    
    CUDNN_CHECK(hipdnnSetConvolution2dDescriptor(conv2_desc, 
        1, 1,    // padding
        1, 1,    // stride
        1, 1,    // dilation
        HIPDNN_CROSS_CORRELATION, 
        HIPDNN_DATA_FLOAT));
    
    // Get Conv2 output dimensions
    CUDNN_CHECK(hipdnnGetConvolution2dForwardOutputDim(conv2_desc, pool1_output_desc, 
        conv2_filter_desc, &conv2_dims.n, &conv2_dims.c, &conv2_dims.h, &conv2_dims.w));
    
    std::cout << "Conv2 output dimensions: " << conv2_dims.n << "x" << conv2_dims.c 
              << "x" << conv2_dims.h << "x" << conv2_dims.w << std::endl;
    
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(conv2_output_desc, HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_FLOAT, conv2_dims.n, conv2_dims.c, conv2_dims.h, conv2_dims.w));
    
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(conv2_bias_desc, HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_FLOAT, 1, conv2_dims.c, 1, 1));

    // Find best algorithm for Conv2
    CUDNN_CHECK(cudnnGetConvolutionForwardAlgorithm_v7(cudnn,
        pool1_output_desc,
        conv2_filter_desc,
        conv2_desc,
        conv2_output_desc,
        requestedAlgoCount,
        &returnedAlgoCount,
        &perfResults));

    // Pool2 setup
    pool2_dims = {conv2_dims.n, conv2_dims.c, conv2_dims.h/2, conv2_dims.w/2};
    
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(pool2_output_desc, HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_FLOAT, pool2_dims.n, pool2_dims.c, pool2_dims.h, pool2_dims.w));

    std::cout << "Pool2 dimensions: " << pool2_dims.n << "x" << pool2_dims.c 
              << "x" << pool2_dims.h << "x" << pool2_dims.w << std::endl;

    // Calculate flattened size for FC1 input
    fc1_input_size = pool2_dims.c * pool2_dims.h * pool2_dims.w;

    // Set up the flattened pool2 descriptor
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(pool2_flat_desc, HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_FLOAT, batch_size, fc1_input_size, 1, 1));

    std::cout << "Pool2 flattened dimensions: " << batch_size << "x" 
              << fc1_input_size << "x1x1" << std::endl;

    // FC1 setup
    CUDNN_CHECK(hipdnnSetFilter4dDescriptor(fc1_weight_desc, HIPDNN_DATA_FLOAT,
        HIPDNN_TENSOR_NCHW, 128, fc1_input_size, 1, 1));
    
    CUDNN_CHECK(hipdnnSetConvolution2dDescriptor(fc1_desc,
        0, 0,    // padding
        1, 1,    // stride
        1, 1,    // dilation
        HIPDNN_CROSS_CORRELATION,
        HIPDNN_DATA_FLOAT));

    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(fc1_output_desc, HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_FLOAT, batch_size, 128, 1, 1));
    
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(fc1_bias_desc, HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_FLOAT, 1, 128, 1, 1));

    // Find best algorithm for FC1
    CUDNN_CHECK(cudnnGetConvolutionForwardAlgorithm_v7(cudnn,
        pool2_flat_desc,
        fc1_weight_desc,
        fc1_desc,
        fc1_output_desc,
        requestedAlgoCount,
        &returnedAlgoCount,
        &perfResults));

    // FC2 setup
    CUDNN_CHECK(hipdnnSetFilter4dDescriptor(fc2_weight_desc, HIPDNN_DATA_FLOAT,
        HIPDNN_TENSOR_NCHW, 10, 128, 1, 1));
    
    CUDNN_CHECK(hipdnnSetConvolution2dDescriptor(fc2_desc,
        0, 0,    // padding
        1, 1,    // stride
        1, 1,    // dilation
        HIPDNN_CROSS_CORRELATION,
        HIPDNN_DATA_FLOAT));

    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(fc2_output_desc, HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_FLOAT, batch_size, 10, 1, 1));
    
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(fc2_bias_desc, HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_FLOAT, 1, 10, 1, 1));

    // Find best algorithm for FC2
    CUDNN_CHECK(cudnnGetConvolutionForwardAlgorithm_v7(cudnn,
        fc1_output_desc,
        fc2_weight_desc,
        fc2_desc,
        fc2_output_desc,
        requestedAlgoCount,
        &returnedAlgoCount,
        &perfResults));

    // ReLU activation setup
    CUDNN_CHECK(hipdnnSetActivationDescriptor(relu_activation,
        HIPDNN_ACTIVATION_RELU,
        HIPDNN_NOT_PROPAGATE_NAN,
        0.0));

    // Calculate workspace sizes for all operations
    size_t workspace_sizes[4];
    CUDNN_CHECK(hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
        input_desc, conv1_filter_desc, conv1_desc, conv1_output_desc,
        HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM, &workspace_sizes[0]));
    
    CUDNN_CHECK(hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
        pool1_output_desc, conv2_filter_desc, conv2_desc, conv2_output_desc,
        HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM, &workspace_sizes[1]));
    
    CUDNN_CHECK(hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
        pool2_flat_desc, fc1_weight_desc, fc1_desc, fc1_output_desc,
        HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM, &workspace_sizes[2]));
    
    CUDNN_CHECK(hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
        fc1_output_desc, fc2_weight_desc, fc2_desc, fc2_output_desc,
        HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM, &workspace_sizes[3]));

    // Find maximum workspace size needed
    workspace_size = *std::max_element(workspace_sizes, workspace_sizes + 4);
    hipMalloc(&d_workspace, workspace_size);

    // Allocate memory for layer outputs
    size_t input_bytes = batch_size * 3 * 32 * 32 * sizeof(float);
    size_t conv1_output_bytes = batch_size * conv1_dims.c * conv1_dims.h * conv1_dims.w * sizeof(float);
    size_t pool1_output_bytes = batch_size * pool1_dims.c * pool1_dims.h * pool1_dims.w * sizeof(float);
    size_t conv2_output_bytes = batch_size * conv2_dims.c * conv2_dims.h * conv2_dims.w * sizeof(float);
    size_t pool2_output_bytes = batch_size * pool2_dims.c * pool2_dims.h * pool2_dims.w * sizeof(float);
    size_t fc1_output_bytes = batch_size * 128 * sizeof(float);
    size_t fc2_output_bytes = batch_size * 10 * sizeof(float);

    hipMalloc(&d_input, input_bytes);
    hipMalloc(&d_conv1_output, conv1_output_bytes);
    hipMalloc(&d_pool1_output, pool1_output_bytes);
    hipMalloc(&d_conv2_output, conv2_output_bytes);
    hipMalloc(&d_pool2_output, pool2_output_bytes);
    hipMalloc(&d_fc1_output, fc1_output_bytes);
    hipMalloc(&d_fc2_output, fc2_output_bytes);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::cerr << "CUDA error after memory allocation: " << hipGetErrorString(error) << std::endl;
        throw std::runtime_error("CUDA memory allocation failed");
    }

    std::cout << "Layer initialization complete." << std::endl;
    std::cout << "Workspace size: " << workspace_size << " bytes" << std::endl;
}

void checkTensorDimensions(hipdnnTensorDescriptor_t desc, const char* name) {
    int n, c, h, w, nStride, cStride, hStride, wStride;
    hipdnnDataType_t dtype;
    CUDNN_CHECK(hipdnnGetTensor4dDescriptor(desc, &dtype, &n, &c, &h, &w,
                              &nStride, &cStride, &hStride, &wStride));
    std::cout << name << " tensor dimensions: " << n << "x" << c << "x" << h << "x" << w << std::endl;
}

void checkFilterDimensions(hipdnnFilterDescriptor_t desc, const char* name) {
    int k, c, h, w;
    hipdnnDataType_t dtype;
    hipdnnTensorFormat_t format;
    CUDNN_CHECK(hipdnnGetFilter4dDescriptor(desc, &dtype, &format, &k, &c, &h, &w));
    std::cout << name << " filter dimensions: " << k << "x" << c << "x" << h << "x" << w << std::endl;
}

void CUDACNNInference::infer(const std::vector<float>& input_data) {
    const float alpha = 1.0f;
    const float beta = 0.0f;
    
    // Verify input data size and copy to device
    size_t expected_input_size = batch_size * 3 * 32 * 32;
    if (input_data.size() != expected_input_size) {
        throw std::runtime_error("Input data size mismatch");
    }

    hipMemcpy(d_input, input_data.data(), input_data.size() * sizeof(float), hipMemcpyHostToDevice);

    // Conv1 layer
    CUDNN_CHECK(hipdnnConvolutionForward(cudnn, &alpha, 
        input_desc, d_input,
        conv1_filter_desc, d_conv1_weight,
        conv1_desc, HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM,
        d_workspace, workspace_size,
        &beta, conv1_output_desc, d_conv1_output));
    
    // Add bias
    CUDNN_CHECK(hipdnnAddTensor(cudnn, 
        &alpha,
        conv1_bias_desc, d_conv1_bias,
        &alpha,  // Important: using alpha here, not beta
        conv1_output_desc, d_conv1_output));
    
    // ReLU
    CUDNN_CHECK(hipdnnActivationForward(cudnn, relu_activation,
        &alpha, conv1_output_desc, d_conv1_output,
        &beta, conv1_output_desc, d_conv1_output));

    // MaxPool1
    CUDNN_CHECK(hipdnnPoolingForward(cudnn, pooling_desc,
        &alpha, conv1_output_desc, d_conv1_output,
        &beta, pool1_output_desc, d_pool1_output));

    // Conv2 layer
    CUDNN_CHECK(hipdnnConvolutionForward(cudnn, &alpha,
        pool1_output_desc, d_pool1_output,
        conv2_filter_desc, d_conv2_weight,
        conv2_desc, HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM,
        d_workspace, workspace_size,
        &beta, conv2_output_desc, d_conv2_output));
    
    // Add bias
    CUDNN_CHECK(hipdnnAddTensor(cudnn, 
        &alpha,
        conv2_bias_desc, d_conv2_bias,
        &alpha,  // Important: using alpha here, not beta
        conv2_output_desc, d_conv2_output));
    
    // ReLU
    CUDNN_CHECK(hipdnnActivationForward(cudnn, relu_activation,
        &alpha, conv2_output_desc, d_conv2_output,
        &beta, conv2_output_desc, d_conv2_output));

    // MaxPool2
    CUDNN_CHECK(hipdnnPoolingForward(cudnn, pooling_desc,
        &alpha, conv2_output_desc, d_conv2_output,
        &beta, pool2_output_desc, d_pool2_output));

    // FC1 layer
    CUDNN_CHECK(hipdnnConvolutionForward(cudnn, &alpha,
        pool2_flat_desc, d_pool2_output,
        fc1_weight_desc, d_fc1_weight,
        fc1_desc, HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
        d_workspace, workspace_size,
        &beta, fc1_output_desc, d_fc1_output));
    
    // Add bias
    CUDNN_CHECK(hipdnnAddTensor(cudnn, 
        &alpha,
        fc1_bias_desc, d_fc1_bias,
        &alpha,  // Important: using alpha here, not beta
        fc1_output_desc, d_fc1_output));
    
    // ReLU
    CUDNN_CHECK(hipdnnActivationForward(cudnn, relu_activation,
        &alpha, fc1_output_desc, d_fc1_output,
        &beta, fc1_output_desc, d_fc1_output));

    // FC2 layer (final layer)
    CUDNN_CHECK(hipdnnConvolutionForward(cudnn, &alpha,
        fc1_output_desc, d_fc1_output,
        fc2_weight_desc, d_fc2_weight,
        fc2_desc, HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
        d_workspace, workspace_size,
        &beta, fc2_output_desc, d_fc2_output));
    
    // Add final bias
    CUDNN_CHECK(hipdnnAddTensor(cudnn, 
        &alpha,
        fc2_bias_desc, d_fc2_bias,
        &alpha,  // Important: using alpha here, not beta
        fc2_output_desc, d_fc2_output));

    // Check for any CUDA errors
    hipError_t cuda_status = hipGetLastError();
    if (cuda_status != hipSuccess) {
        throw std::runtime_error(std::string("CUDA error during inference: ") + 
                               hipGetErrorString(cuda_status));
    }
}


std::vector<float> CUDACNNInference::getOutput() {
    std::vector<float> output(10);
    
    // Copy the output from device to host
    hipError_t status = hipMemcpy(output.data(), d_fc2_output, 
                                   output.size() * sizeof(float), 
                                   hipMemcpyDeviceToHost);
    
    if (status != hipSuccess) {
        throw std::runtime_error(std::string("Failed to copy output from device: ") + 
                               hipGetErrorString(status));
    }
    
    // Apply softmax normalization
    float max_val = *std::max_element(output.begin(), output.end());
    float sum = 0.0f;
    
    // Subtract max for numerical stability and compute exp
    for (float& val : output) {
        val = std::exp(val - max_val);
        sum += val;
    }
    
    // Normalize
    for (float& val : output) {
        val /= sum;
    }
    
    return output;
}

int main() {
    try {
        std::cout << "Loading validation data..." << std::endl;
        auto validation_images = loadBinaryFile<float>("../../../data/validation/validation_images.bin");
        auto validation_labels = loadBinaryFile<int>("../../../data/validation/validation_labels.bin");

        // Original image size for CIFAR-10 (3 channels, 32x32 resolution)
        size_t image_size = 3 * 32 * 32;

        // Organize the original data into individual images
        std::vector<std::vector<float>> images;
        for (size_t i = 0; i < validation_images.size(); i += image_size) {
            images.push_back(std::vector<float>(validation_images.begin() + i, 
                                              validation_images.begin() + i + image_size));
        }

        // Repeat the dataset to increase the total number of images
        int repeat_factor = 10; // Adjust this factor as needed to increase the dataset size
        std::vector<std::vector<float>> repeated_images;
        std::vector<int> repeated_labels;

        for (int i = 0; i < repeat_factor; ++i) {
            repeated_images.insert(repeated_images.end(), images.begin(), images.end());
            repeated_labels.insert(repeated_labels.end(), validation_labels.begin(), validation_labels.end());
        }

        int total_images = repeated_images.size();

        std::cout << "Total images after repeating: " << total_images << std::endl;

        std::cout << "Creating CUDA inference engine..." << std::endl;
        CUDACNNInference cnn;

        std::cout << "\n=== Starting Evaluation ===" << std::endl;
        std::cout << "Model type: CUDA Core" << std::endl;

        // Create CUDA events for timing
        hipEvent_t start, stop;
        CUDA_CHECK(hipEventCreate(&start));
        CUDA_CHECK(hipEventCreate(&stop));

        size_t correct_count = 0;
        float total_time = 0.0f;

        // Warmup run
        std::cout << "Performing warmup runs..." << std::endl;
        for (int i = 0; i < 10; i++) {
            cnn.infer(repeated_images[0]);
        }

        // Main evaluation loop
        std::cout << "Starting main evaluation..." << std::endl;
        for (size_t i = 0; i < total_images; ++i) {
            try {
                CUDA_CHECK(hipEventRecord(start));
                
                cnn.infer(repeated_images[i]);
                std::vector<float> output = cnn.getOutput();
                
                CUDA_CHECK(hipEventRecord(stop));
                CUDA_CHECK(hipEventSynchronize(stop));
                
                float milliseconds = 0;
                CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
                total_time += milliseconds;

                int predicted_label = std::distance(output.begin(), 
                                                    std::max_element(output.begin(), output.end()));
                
                if (predicted_label == repeated_labels[i]) {
                    ++correct_count;
                }

                // if (i % 100 == 0) {

                //     // Print running statistics (debugging)
                //     // float running_accuracy = (static_cast<float>(correct_count) / (i + 1)) * 100.0f;
                //     // std::cout << "\nProcessed " << i + 1 << "/" << total_images << " images" << std::endl;
                //     // std::cout << "Running accuracy: " << std::fixed << std::setprecision(2) 
                //     //           << running_accuracy << "%" << std::endl;
                //     // std::cout << "Current inference time: " << std::fixed << std::setprecision(3) 
                //     //           << milliseconds << " ms" << std::endl;
                    
                //     // Print top 5 predictions for current image (debugging)
                //     // std::vector<std::pair<int, float>> scores;
                //     // for (size_t j = 0; j < output.size(); ++j) {
                //     //     scores.emplace_back(j, output[j]);
                //     // }
                //     // std::sort(scores.begin(), scores.end(),
                //     //           [](const auto& a, const auto& b) { return a.second > b.second; });
                    
                //     // std::cout << "Top 5 predictions for current image:" << std::endl;
                //     // for (int k = 0; k < std::min(5, static_cast<int>(scores.size())); ++k) {
                //     //     std::cout << "  Class " << std::setw(2) << scores[k].first 
                //     //               << ": " << std::fixed << std::setprecision(4) 
                //     //               << (scores[k].second * 100.0f) << "%" << std::endl;
                //     // }
                //     // std::cout << "True label: " << repeated_labels[i] << std::endl;
                // }
            }
            catch (const std::exception& e) {
                std::cerr << "Error processing image " << i << ": " << e.what() << std::endl;
                continue;
            }
        }

        // Print final statistics
        float accuracy = static_cast<float>(correct_count) / total_images * 100.0f;
        float avg_time = total_time / total_images;
        float throughput = 1000.0f / avg_time;

        std::cout << "\n=== Final Results ===" << std::endl;
        std::cout << "Model type: CUDA Core" << std::endl;
        std::cout << "Total images: " << total_images << std::endl;
        std::cout << "Correct predictions: " << correct_count << std::endl;
        std::cout << "Accuracy: " << std::fixed << std::setprecision(2) << accuracy << "%" << std::endl;
        std::cout << "Average inference time: " << std::fixed << std::setprecision(3) 
                  << avg_time << " ms" << std::endl;
        std::cout << "Throughput: " << std::fixed << std::setprecision(1) 
                  << throughput << " images/second" << std::endl;
        std::cout << "Total evaluation time: " << std::fixed << std::setprecision(2) 
                  << total_time / 1000.0f << " seconds" << std::endl;

        CUDA_CHECK(hipEventDestroy(start));
        CUDA_CHECK(hipEventDestroy(stop));

    } catch (const std::exception& e) {
        std::cerr << "Fatal error: " << e.what() << std::endl;
        return 1;
    }

    return 0;
}